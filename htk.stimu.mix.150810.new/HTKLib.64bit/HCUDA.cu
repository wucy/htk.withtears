#include "hip/hip_runtime.h"
/* ----------------------------------------------------------- */
/*                                                             */
/*                          ___                                */
/*                       |_| | |_/   SPEECH                    */
/*                       | | | | \   RECOGNITION               */
/*                       =========   SOFTWARE                  */
/*                                                             */
/*                                                             */
/* ----------------------------------------------------------- */
/*         Copyright: Microsoft Corporation                    */
/*          1995-2000 Redmond, Washington USA                  */
/*                    http://www.microsoft.com                 */
/*                                                             */
/*   Use of this software is governed by a License Agreement   */
/*    ** See the file License for the Conditions of Use  **    */
/*    **     This banner notice must not be removed      **    */
/*                                                             */
/* ----------------------------------------------------------- */
/*                 File: HCUDA.cu:   CUDA Utilities            */
/* ----------------------------------------------------------- */

#ifdef __cplusplus
extern "C" {
#endif

char *hcuda_version = "!HVER!HCUDA:   3.4.1 [CUED 30/11/13]";
char *hcuda_vc_id = "$Id: HCUDA.cu,v 1.1.1.1 2013/11/13 09:54:58 cz277 Exp $";

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "HCUDA.h"
#include "HShell.h"
#include "HMem.h"
#include "HMath.h"
#include "cfgs.h"


/* --------------------------- Trace Flags ------------------------ */

#define CEIL(x,y) (((x)+(y)-1) / (y))

/* --------------------------- Trace Flags ------------------------ */

static int trace = 0;                           /*  */
#define T_TOP 0001                              /* Top Level tracing */

static ConfParam *cParm[MAXGLOBS];              /* config parameters */
static int nParm = 0;

static int GPUDevId = -1;                       /*  */
static Boolean GPUInit = FALSE;                 /*  */
static char *GPUIdEnvVar = "";                  /*  */
hipblasHandle_t handle;				/*  */
static size_t GPUMemUsed = 0;			/*  */

/* ----------------------- Device Management ---------------------- */

/*  */
static void ShowAllGPUs(void) {
    int nGPU, i;
    hipError_t error;
    hipDeviceProp_t prop;
    /*CUResult result;*/

    error = hipGetDeviceCount(&nGPU);    
    if (error != hipSuccess) {
        HError(9999, "ShowAllGPUs: %s", hipGetErrorString(error)); 
    }
    if (nGPU == 0) {
        HError(9999, "ShowAllGPUs: No GPU device");
    }
    for (i = 0; i < nGPU; ++i) {
        error = hipGetDeviceProperties(&prop, i);
        if (error != hipSuccess) {
            HError(9999, "ShowAllGPUs: %s", hipGetErrorString(error));
        }
        printf("GPU %d: %s, %dMB, SM = %d.%d", i, prop.name, prop.totalGlobalMem / 1048576, prop.major, prop.minor);
        if (GPUDevId == i)
            printf(" [Selected]");
        printf("\n");
    }
}

/* To check CUDA requirement */
static void CheckCUDAReq(hipDeviceProp_t *prop)
{
    int driverVer;
    int runtimeVer;
    int cublasVer;
    hipError_t error;    
    hipblasStatus_t status;
    
    error = hipDriverGetVersion(&driverVer);
    if (error != hipSuccess) {
        HError(9999, "CheckCUDAReq: %s", hipGetErrorString(error));
    }
    if (driverVer < MINCUDAVER) {
        HError(9999, "CheckCUDAReq: CUDA driver version %d is lower than the minimum required version %d", driverVer, MINCUDAVER);
    }

    error = hipRuntimeGetVersion(&runtimeVer);
    if (error != hipSuccess) {
        HError(9999, "CheckCUDAReq: %s", hipGetErrorString(error));
    }
    if (runtimeVer < MINCUDAVER) {
        HError(9999, "CheckCUDAReq: CUDA runtime version %d is lower than the minimum required version %d", runtimeVer, MINCUDAVER);
    }

    status = cublasGetVersion(handle, &cublasVer);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(9999, "CheckCUDAReq: Fail to get CUBLAS library version");
    }
    if (cublasVer < MINCUDAVER) {
        HError(9999, "CheckCUDAReq: CUBLAS library version %d is lower than the minimum required version %d", cublasVer, MINCUDAVER);
    }

    if (prop->major <= MINMAJORSMARCH && prop->minor <= MINMINORSMARCH) {
        HError(9999, "CheckCUDAReq: SM architecture is lower than the minimum requirement, %d.%d", MINMAJORSMARCH, MINMINORSMARCH);
    }

    printf("CUDA driver version %d\n", driverVer);
    printf("CUDA runtime version %d\n", runtimeVer);
    printf("CUBLAS library version %d\n", cublasVer);
}

/* Initialize the GPU device. It first loads the GPU device
   from the config file. Then
*/
void InitCUDA(void)
{
    int intVal;
    char buf[256];
    ConfParam *cpVal;

    Register(hcuda_version, hcuda_vc_id);

    /* load parameters from the config file */
    nParm = GetConfig("HCUDA", TRUE, cParm, MAXGLOBS);
    if (nParm > 0) {
        if (GetConfInt(cParm, nParm, "TRACE", &intVal)) {
            trace = intVal;
        }
        if (GetConfAny(cParm, nParm, "GPUID", &cpVal)) {
            if (cpVal->kind == IntCKind) {
                GPUDevId = cpVal->val.i;
            }
            else if (cpVal->kind == StrCKind) {
                strcpy(buf, cpVal->val.s);
                GPUIdEnvVar = (char *) New(&gcheap, sizeof(char) * strlen(buf));
                strcpy(GPUIdEnvVar, buf);
            }
            else {
                HError(9999, "InitCUDA: Unknown GPUID kind");
            }
        }
    }
}

/*  */
void StartCUDA(void) {
    char *envVar;
    hipError_t error;
    hipblasStatus_t status;
    hipDeviceProp_t prop;

    /* initialize the library and device */
    if (!GPUInit) {
        /* select a device */
        if (strcmp(GPUIdEnvVar, "") != 0) { /* use env variable */
            envVar = getenv(GPUIdEnvVar);
            if (envVar == NULL) {
                printf("InitCUDA: Fail to get environment variable %s\n", GPUIdEnvVar);
            }
            GPUDevId =  atoi(envVar);
        }
        if (GPUDevId < 0) {
            error = hipChooseDevice(&GPUDevId, &prop);
            if (error != hipSuccess) {
                HError(9999, "InitCUDA: %s", hipGetErrorString(error));
            }
        }
        error = hipSetDevice(GPUDevId);
        if (error != hipSuccess) {
            HError(9999, "InitCUDA: %s", hipGetErrorString(error));
        }
        error = hipGetDeviceProperties(&prop, GPUDevId);
        if (error != hipSuccess) {
            HError(9999, "InitCUDA: %s", hipGetErrorString(error));
        }
        /* initiate CUBLAS */
        status = hipblasCreate(&handle);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            HError(9999, "InitCUDA: Fail to initialise CUBLAS");
        }
        /* check version */
        CheckCUDAReq(&prop);
        /* set GPUInit flag */
        GPUInit = TRUE;
        /* show devices */
        ShowAllGPUs();
    }
    else {
        printf("InitCUDA: GPU device %d already initialised", GPUDevId);
    }
}

/*  */
void StopCUDA(void) {
    if (GPUInit) {
        /* destroy the context on the GPU */
        hipblasDestroy(handle);
        /* shutdown CUBLAS */
        hipDeviceReset();
        /* reset GPU IDs and the flag */
        GPUDevId = -1;
        GPUInit = FALSE;
    }
    else {
        printf("StopCUDA: GPU device has already stopped");
    }
}

/* --------------------------- Trace Flags ------------------------ */

__global__ void HKern_SetNSegment(NFloat val, NFloat *segPtr, int segLen) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        segPtr[pos] = val;
    }
}

__global__ void HKern_ScaledSelfAddNSegment(NFloat *rhPtr, int segLen, NFloat scale, NFloat *lhPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        lhPtr[pos] = scale * lhPtr[pos] + rhPtr[pos];
    }
}

__global__ void HKern_DupNSegment(NFloat *srcPtr, int segLen, NFloat *dstPtr, int times) {
    int srcPos, dstPos;
    
    dstPos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (dstPos < segLen * times) {
        srcPos = dstPos % segLen;
        dstPtr[dstPos] = srcPtr[srcPos];
    }
}

__global__ void HKern_SubNSegment(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        resPtr[pos] = lhPtr[pos] - rhPtr[pos];
    }
}

__global__ void HKern_MulNSegment(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        resPtr[pos] = lhPtr[pos] * rhPtr[pos];
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyAffineAct(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        dstPtr[pos] = scalePtr[colIdx] * srcPtr[pos] + shiftPtr[colIdx];
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyDAffineAct(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        dstPtr[pos] = scalePtr[colIdx];
    }
}


/* cz277 - pact */
__global__ void HKern_ApplyTrAffineAct(NFloat *errPtr, NFloat *actPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, Boolean accFlag, NFloat *dScalePtr, NFloat *dShiftPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step, off = THREADPERBLOCK;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;		/* dScale */
        tmpPtr[off + thdIdx] = 0.0;	/* dShift */
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += errPtr[pos] * actPtr[pos];
            tmpPtr[off + thdIdx] += errPtr[pos];
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                    tmpPtr[off + thdIdx] += tmpPtr[off + pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE) {
                dScalePtr[colIdx] = 0.0;
                dShiftPtr[colIdx] = 0.0;
            }
            dScalePtr[colIdx] += tmpPtr[0];
            dShiftPtr[colIdx] += tmpPtr[off + 0];
        }
    }
}

/* cz277 - laf */
__global__ void HKern_AccMeanNSegment(NFloat *valPtr, int row, int col, NFloat tSamp, NFloat *meanPtr) {
        extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;/*srcPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += valPtr[pos] / tSamp;
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            meanPtr[colIdx] += tmpPtr[0];
        }
    }
}

/* cz277 - laf */
__global__ void HKern_AccVarianceNSegment(NFloat *valPtr, int row, int col, NFloat tSamp, NFloat *meanPtr, NFloat *varPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;/*srcPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += pow(valPtr[pos] - meanPtr[colIdx], 2) / tSamp;
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            varPtr[colIdx] += tmpPtr[0];
        }
    }
}


/* cz277 - pact */
__global__ void HKern_ApplyParmReLUAct(NFloat *srcPtr, int row, int col, NFloat *posPtr, NFloat *negPtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        if (srcPtr[pos] > 0.0)
            dstPtr[pos] = posPtr[colIdx] * srcPtr[pos];
        else
            dstPtr[pos] = negPtr[colIdx] * srcPtr[pos];
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyDParmReLUAct(NFloat *inpPtr, int row, int col, NFloat *posPtr, NFloat *negPtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        if (inpPtr[pos] > 0.0)
            dstPtr[pos] = posPtr[colIdx];
        else
            dstPtr[pos] = negPtr[colIdx];
    }
}


/* cz277 - pact */
__global__ void HKern_ApplyTrParmReLUAct(NFloat *errPtr, NFloat *inpPtr, int row, int col, Boolean accFlag, NFloat *dPosPtr, NFloat *dNegPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step, off = THREADPERBLOCK;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;		/* alpha */
        tmpPtr[off + thdIdx] = 0.0;	/* beta */
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            if (inpPtr[pos] > 0.0)
                tmpPtr[thdIdx] += errPtr[pos] * inpPtr[pos];
            else
                tmpPtr[off + thdIdx] += errPtr[pos] * inpPtr[pos];
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                    tmpPtr[off + thdIdx] += tmpPtr[off + pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE) {
                dPosPtr[colIdx] = 0.0;
                dNegPtr[colIdx] = 0.0;
            }
            dPosPtr[colIdx] += tmpPtr[0];
            dNegPtr[colIdx] += tmpPtr[off + 0];
        }
    }
}


/* cz277 - laf */
__global__ void HKern_ApplyPReLUAct(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        if (srcPtr[pos] > 0.0)
            dstPtr[pos] = scalePtr[colIdx] * srcPtr[pos];
        else
            dstPtr[pos] = 0.0;
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyDPReLUAct(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *dstPtr) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    rowIdx = pos / col;
    colIdx = pos % col;
    if (rowIdx < row) {
        if (scalePtr[colIdx] != 0.0 && srcPtr[pos] / scalePtr[colIdx] > 0.0)
            dstPtr[pos] = scalePtr[colIdx];
        else
            dstPtr[pos] = 0.0;
    }
}

/* cz277 - pact */
__global__ void HKern_ApplyTrPReLUAct(NFloat *errPtr, NFloat *srcPtr, int row, int col, NFloat *scalePtr, Boolean accFlag, NFloat *dScalePtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;
    NFloat act;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;	/*srcPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            if (scalePtr[colIdx] != 0.0) {
                act = srcPtr[pos] / scalePtr[colIdx];
                if (act > 0.0)
                    tmpPtr[thdIdx] += errPtr[pos] * act;
            }
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE)
                dScalePtr[colIdx] = 0.0;
            dScalePtr[colIdx] += tmpPtr[0];
        }
    }
}

__global__ void HKern_ApplyReLUAct(NFloat *srcPtr, int len, NFloat scale, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        if (srcPtr != dstPtr && srcPtr[pos] > 0) {
            dstPtr[pos] = srcPtr[pos];
        }
        if (srcPtr[pos] < 0) {
            dstPtr[pos] = srcPtr[pos] * scale;
            /* cz277 - standard ReLU */
            /*dstPtr[pos] = 0.0;*/
        }
    }
}

__global__ void HKern_ApplyDReLUAct(NFloat *srcPtr, int len, NFloat scale, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        if (srcPtr[pos] > 0.0) {
            dstPtr[pos] = 1.0;
        }
        else {
            dstPtr[pos] = scale;
            /* cz277 - standard ReLU */
            /*dstPtr[pos] = 0.0;*/
        }
    }
}

__global__ void HKern_ApplyDLinearAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = 1.0;
    }
}

__global__ void HKern_ApplyLHUCSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *rolePtr, NFloat *dstPtr) {
    int pos, colIdx;
    NFloat floatVal, lhucVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        floatVal = -1.0 * rolePtr[colIdx];
        CHKNFLTEXPE(floatVal)
        lhucVal = 2.0 / (1.0 + exp(floatVal));
        floatVal = -1.0 * srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = lhucVal * 1.0 / (1.0 + exp(floatVal));
    }
}

__global__ void HKern_ApplyDLHUCSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *rolePtr, NFloat *dstPtr) {
    int pos, colIdx;
    NFloat floatVal, lhucVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        floatVal = -1.0 * rolePtr[colIdx];
        CHKNFLTEXPE(floatVal)
        lhucVal = 2.0 / (1.0 + exp(floatVal));
        floatVal = srcPtr[pos] / lhucVal;
        dstPtr[pos] = srcPtr[pos] * (1.0 - floatVal);
    }
}

__global__ void HKern_ApplyTrLHUCSigmoidActCUDA(NFloat *errPtr, NFloat *actPtr, int row, int col, NFloat *rolePtr, Boolean accFlag, NFloat *dRolePtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;
    NFloat floatVal;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        floatVal = -1.0 * rolePtr[colIdx];
        CHKNFLTEXPE(floatVal)
        floatVal = 0.5 * 2.0 / (1.0 + exp(floatVal));
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;/*actPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += errPtr[pos] * actPtr[pos] * (1.0 - floatVal);
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE)
                dRolePtr[colIdx] = 0.0;
            dRolePtr[colIdx] += tmpPtr[0];
        }
    }
}


__global__ void HKern_ApplyParmSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat* thetaPtr, NFloat *dstPtr) {
    int pos, colIdx;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        floatVal = (-1.0) * gammaPtr[colIdx] * srcPtr[pos] + thetaPtr[colIdx];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = etaPtr[colIdx] / (1.0 + exp(floatVal));
    }
}

__global__ void HKern_ApplyDParmSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, NFloat *dstPtr) {
    int pos, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        if (etaPtr[colIdx] != 0.0)
            dstPtr[pos] = gammaPtr[colIdx] * srcPtr[pos] * (1.0 - srcPtr[pos] / etaPtr[colIdx]);
        else
            dstPtr[pos] = 0.0;
    }
}

__global__ void HKern_ApplyTrParmSigmoidActCUDA(NFloat *errPtr, NFloat *inpPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, Boolean accFlag, NFloat *dEtaPtr, NFloat *dGammaPtr, NFloat *dThetaPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step, off = THREADPERBLOCK;
    NFloat floatVal, fracVal;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;	/*actPtr[base + idx * col];*/
        tmpPtr[off + thdIdx] = 0.0;
        tmpPtr[off + off + thdIdx] = 0.0;
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            floatVal = (-1.0) * gammaPtr[colIdx] * inpPtr[pos] + thetaPtr[colIdx];
            CHKNFLTEXPE(floatVal)
            fracVal = 1.0 / (1.0 + exp(floatVal));
            tmpPtr[thdIdx] += errPtr[pos] * fracVal;
            if (etaPtr[colIdx] != 0.0) {
                tmpPtr[off + thdIdx] += errPtr[pos] * inpPtr[pos] * etaPtr[colIdx] * fracVal * (1.0 - fracVal);
                tmpPtr[off + off + thdIdx] -= errPtr[pos] * etaPtr[colIdx] * fracVal * (1.0 - fracVal);
            }  
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                    tmpPtr[off + thdIdx] += tmpPtr[off + pos];
                    tmpPtr[off + off + thdIdx] += tmpPtr[off + off + pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE) {
                dEtaPtr[colIdx] = 0.0;
                dGammaPtr[colIdx] = 0.0;
                dThetaPtr[colIdx] = 0.0;
            }
            dEtaPtr[colIdx] += tmpPtr[0];
            dGammaPtr[colIdx] += tmpPtr[off + 0];
            dThetaPtr[colIdx] += tmpPtr[off + off + 0];
        }
    }
}


__global__ void HKern_ApplyPSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *dstPtr) {
    int pos, colIdx;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        floatVal = (-1.0) * srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = etaPtr[colIdx] / (1.0 + exp(floatVal));
    }
}

__global__ void HKern_ApplyDPSigmoidAct(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *dstPtr) {
    int pos, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        colIdx = pos % col;
        /* dstPtr[pos] = srcPtr[pos] * (1.0 - srcPtr[pos] / etaPtr[colIdx]); */
        if (etaPtr[colIdx] != 0.0)
            dstPtr[pos] = 1.0 / etaPtr[colIdx] * srcPtr[pos] * (etaPtr[colIdx] - srcPtr[pos]);
        else
            dstPtr[pos] = 0.0;
    }
}

__global__ void HKern_ApplyTrPSigmoidActCUDA(NFloat *errPtr, NFloat *srcPtr, NFloat *etaPtr, int row, int col, Boolean accFlag, NFloat *dEtaPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;	/*actPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            /* tmpPtr[thdIdx] += errPtr[pos] * srcPtr[pos] / etaPtr[colIdx]; */
            if (etaPtr[colIdx] != 0.0)
                tmpPtr[thdIdx] += errPtr[pos] * 1.0 / etaPtr[colIdx] * srcPtr[pos];
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE)
                dEtaPtr[colIdx] = 0.0;
            dEtaPtr[colIdx] += tmpPtr[0];
        }
    }
}


__global__ void HKern_ApplySigmoidAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = -1.0 * srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = 1.0 / (1.0 + exp(floatVal));
    }
}

__global__ void HKern_ApplyDSigmoidAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = (1 - srcPtr[pos]) * srcPtr[pos];
    }
}

__global__ void HKern_ApplyTanHAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        floatVal = exp(floatVal);
        dstPtr[pos] = (floatVal - 1.0 / floatVal) / (floatVal + 1.0 / floatVal);
    }
}

__global__ void HKern_ApplyDTanHAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = 1 - pow(srcPtr[pos], 2);
    }
}

__global__ void HKern_DualSumByRow(NFloat *srcPtr, int col, int size, int incr, NFloat *dstPtr) {
    int lhpos, rhpos, lhidx, rhidx, mod;

    lhpos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (lhpos < size) {
        mod = incr * 2;
        lhidx = lhpos % col;
        if (lhidx % mod == 0) {
            rhidx = lhidx + incr;
            rhpos = lhpos + incr;
            if (rhidx >= col) {
                dstPtr[lhpos] = srcPtr[lhpos];
            }
            else {
                dstPtr[lhpos] = srcPtr[lhpos] + srcPtr[rhpos];
            }
        }
    }
}

__global__ void HKern_ApplySoftmaxAct(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int frame, i, base, off;
    NFloat den, floatVal;

    frame = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (frame < row) {
        den = 0.0;
        base = frame * col;
        for (i = 0, off = base; i < col; ++i, ++off) {
            floatVal = srcPtr[off];
            CHKNFLTEXPE(floatVal)
            floatVal = exp(floatVal);
            dstPtr[off] = floatVal;
            den += floatVal;
        }
        for (i = 0, off = base; i < col; ++i, ++off) {
            dstPtr[off] /= den;
        }
    }
}


//cw564 - stimu -- begin
#define LITTLE 0.0000001
#define RESVAR 9.5
#define SCALERVAR 100.0
#define COMBLITTLE 1e-6

__device__ inline NFloat calc_tanh(NFloat raw) {
    float floatVal = raw;
    CHKNFLTEXPE(floatVal)
    floatVal = exp(floatVal);
    return (floatVal - 1.0 / floatVal) / (floatVal + 1.0 / floatVal);
}

__device__ inline NFloat calc_exp(NFloat raw) {
    CHKNFLTEXPE(raw)
    return exp(raw);
}


__global__ void HKern_MixValidate(NFloat *act, int num_phones, int len) {
    int myidx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (myidx >= 2 * num_phones) {
        return;
    }
    int my_ptr = myidx / 2 * 6 + myidx % 2;
    if (act[my_ptr] > 1.0) {
        act[my_ptr] = 1.0;
    }
    else if (act[my_ptr] < 0.0){
        act[my_ptr] = 0.0;
    }
    /*if (myidx >= len) return;
    int my_dim = myidx % 5;
    if (my_dim == 4) {
        my_dim = 5;    
    }
    int my_phone = myidx / 5;
    int my_ptr = my_dim + my_phone * 6;
    if (my_dim == 0 || my_dim == 1) {
        return;
    }
    else if (my_dim == 3) {
        if (act[my_ptr] <= -1.0) {
            act[my_ptr] = -0.999999;
        }
        else if (act[my_ptr] >= 1.0) {
            act[my_ptr] = 0.999999;
        }
    }
    else if (my_dim == 2 || my_dim == 5) {
        if (act[my_ptr] <= 0) {
            act[my_ptr] = 0.000001;
        }
    }*/
}

void MixValidateCUDA(NFloat *act, int num_phones, int len) {
    int blocks = CEIL(2 * num_phones, THREADPERBLOCK);
    //printf("%d %d\n", blocks, num_phones);exit(0);
    HKern_MixValidate<<< blocks, THREADPERBLOCK >>>(act, num_phones, len);
}


__global__ void HKern_CalcMixSurface(NFloat* mix_surface, NFloat* actParmVec, int comb_dim, int mixnodes, float grid_var, int tot, int grid_one_dim) {

    int i, q, r, row_no, col_no;
    float x, y, norm, s1, s2, rou, s1s2, one_min_rousqr;
    float diff1, diff2, diff;
    i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < tot) {
        q = i / comb_dim;
        r = i % comb_dim;
        row_no = r / grid_one_dim;
        col_no = r % grid_one_dim;
        x = (0.5 + col_no) / grid_one_dim;
        y = (0.5 + row_no) / grid_one_dim;
    
        s1 = calc_exp(actParmVec[q * 6 + 2]);
        s2 = calc_exp(actParmVec[q * 6 + 5]);
        rou = calc_tanh(actParmVec[q * 6 + 3]);

        //s1 = actParmVec[q * 6 + 2];
        //s2 = actParmVec[q * 6 + 5];
        //rou = actParmVec[q * 6 + 3];

        s1s2 = s1 * s2;

        one_min_rousqr = 1 - rou * rou;
        norm = 1.0 / (PI * 2.0 * sqrtf(one_min_rousqr) * s1s2);

        diff1 = x - actParmVec[q * 6];
        diff2 = y - actParmVec[q * 6 + 1];
        diff = -(powf(diff1 / s1, 2) + powf(diff2 / s2, 2) - 2 * rou * diff1 * diff2 / s1s2) / (2 * one_min_rousqr);
        
        mix_surface[i] = exp(diff) * norm;
    }
}

void CalcMixSurfaceCUDA(NFloat* mix_surface, NFloat* actParmVec, int comb_dim, int mixnodes, float grid_var) {
    int tot = comb_dim * mixnodes;
    //printf("%d %d\n", comb_dim, mixnodes);
    int nBlocks = CEIL(tot, THREADPERBLOCK);
    int grid_one_dim = (int)sqrtf(comb_dim);
    HKern_CalcMixSurface<<< nBlocks, THREADPERBLOCK >>>(mix_surface, actParmVec, comb_dim, mixnodes, grid_var, tot, grid_one_dim);
}

__global__ void HKern_ResetZero(NFloat *dact, int len) {
    int my = blockIdx.x * blockDim.x + threadIdx.x;
    if (my >= len) return;
    dact[my] = 0.0;
}

__global__ void HKern_ApplyGradMixRels(NFloat* dact, NFloat* act, NFloat* dyFeaMat, NFloat* mix_surface, NFloat * raw_y, int batLen, int nodeNum, int comb_dim, int mixnodes, NFloat mix_rel_l2_penalty) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, batIdx, mixIdx, mixdimIdx, thdNum, dybase, mixbase, idx, incr, pos, nodeIdx;
    int total_jobs = batLen * comb_dim;
    NFloat sumVal, tmpVal, x, y, s1, s2, rou, m1, m2;
    int row_no, col_no;
    thdIdx = threadIdx.x;
    mixIdx = blockIdx.x;
    mixdimIdx = 3;
    thdNum = min(blockDim.x, total_jobs);
    if (thdIdx < thdNum && mixIdx < mixnodes) { // && (mixIdx == 1 || mixIdx  == 10)) {
        idx = thdIdx;
        tmpPtr[thdIdx] = 0;

        m1 = act[mixIdx * 6 + 0];
        m2 = act[mixIdx * 6 + 1];
        s1 = calc_exp(act[mixIdx * 6 + 2]);
        rou= calc_tanh(act[mixIdx * 6 + 3]);
        s2 = calc_exp(act[mixIdx * 6 + 5]);
        while (idx < total_jobs) {
            batIdx = idx / comb_dim;
            nodeIdx = idx % comb_dim;
            row_no = nodeIdx / 32;
            col_no = nodeIdx % 32;

            x = (0.5 + col_no) / 32.0;
            y = (0.5 + row_no) / 32.0;
            tmpVal =  rou / powf(1-rou, 2) + 
                    (y * rou * s1 - x * s2 - rou * m2 * s1 + m1 * s2) * (-y * s1 + x * rou * s2 - rou * m1 * s2 + m2 * s1) / (powf(s1, 2) * powf(s2, 2) * powf(rou - 1, 2) * powf(rou + 1, 2));
                            //scaler                                        * mixweights                                  * comb_dy
            tmpPtr[thdIdx] += raw_y[batIdx * nodeNum + comb_dim + mixnodes] * raw_y[batIdx * nodeNum + comb_dim + mixIdx] * dyFeaMat[batIdx * comb_dim + nodeIdx] 
                    //mix_dist
                    * mix_surface[mixIdx * comb_dim + nodeIdx] * tmpVal;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++ incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        if (thdIdx == 0) {
            //dact[mixIdx * 6 + mixdimIdx] = 2.0 * sumVal / comb_dim * 10.0;
            dact[mixIdx * 6 + mixdimIdx] = sumVal * (1 - powf(rou, 2)) + act[mixIdx * 6 + mixdimIdx] * mix_rel_l2_penalty;
            //dact[mixIdx * 6 + mixdimIdx] = 0;
        }
    }
}

void ApplyGradMixRelsCUDA(NFloat* dact, NFloat* act, NFloat* dyFeaMat, NFloat* mix_surface, NFloat* raw_y, int batLen, int nodeNum, int comb_dim, int mixnodes, NFloat mix_rel_l2_penalty) {
    int blocks = mixnodes;
    //int threads = batLen;
    int sbytes = THREADPERBLOCK * sizeof(float);
    int zblocks = CEIL(comb_dim, THREADPERBLOCK);
    //HKern_ResetZero<<< zblocks, THREADPERBLOCK >>>(dact, comb_dim);
    HKern_ApplyGradMixRels<<< blocks, THREADPERBLOCK, sbytes >>>(dact, act, dyFeaMat, mix_surface, raw_y, batLen, nodeNum, comb_dim, mixnodes, mix_rel_l2_penalty);
}



__global__ void HKern_ApplyGradMixVars(NFloat* dact, NFloat* act, NFloat* dyFeaMat, NFloat* mix_surface, NFloat * raw_y, int batLen, int nodeNum, int comb_dim, int mixnodes, NFloat mix_var_l2_penalty, NFloat gridvar) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, batIdx, mixIdx, mixdimIdx, thdNum, dybase, mixbase, idx, incr, pos, nodeIdx;
    int total_jobs = batLen * comb_dim;
    NFloat sumVal, tmpVal, x, y, s1, s2, rou, m1, m2;
    int row_no, col_no;
    thdIdx = threadIdx.x;
    mixIdx = blockIdx.x / 2;
    mixdimIdx = blockIdx.x % 2;
    if (mixdimIdx) {
        mixdimIdx = 5; //second var
    } else {
        mixdimIdx = 2; //first var
    }
    thdNum = min(blockDim.x, total_jobs);
    if (thdIdx < thdNum && mixIdx < mixnodes) { // && (mixIdx == 1 || mixIdx  == 10)) {
        idx = thdIdx;
        tmpPtr[thdIdx] = 0;

        m1 = act[mixIdx * 6 + 0];
        m2 = act[mixIdx * 6 + 1];
        s1 = calc_exp(act[mixIdx * 6 + 2]);
        rou= calc_tanh(act[mixIdx * 6 + 3]);
        s2 = calc_exp(act[mixIdx * 6 + 5]);
        while (idx < total_jobs) {
            batIdx = idx / comb_dim;
            nodeIdx = idx % comb_dim;
            row_no = nodeIdx / 32;
            col_no = nodeIdx % 32;

            x = (0.5 + col_no) / 32.0;
            y = (0.5 + row_no) / 32.0;
            if (mixdimIdx == 2) {
                //tmpVal =  -1.0 / (s1) + (x - m1) * (y * rou * s1 + s2 * m1 - s2 * x - rou * m2 * s1) / ((powf(rou, 2) - 1) * powf(s1, 3) * s2);
                tmpVal =  -1.0 / (s1) + (m1 - x) * (-m1 * s2 + m2 * rou * s1 - rou * s1 * y + s2 * x) / ((powf(rou, 2) - 1) * powf(s1, 3) * s2);
                                //scaler                                        * mixweights                                  * comb_dy
                tmpPtr[thdIdx] += raw_y[batIdx * nodeNum + comb_dim + mixnodes] * raw_y[batIdx * nodeNum + comb_dim + mixIdx] * dyFeaMat[batIdx * comb_dim + nodeIdx] 
                        //mix_dist
                        * mix_surface[mixIdx * comb_dim + nodeIdx] * tmpVal;
            }
            else {
                //tmpVal =  -1.0 / (s2) + (y - m2) * (x * rou * s2 + s1 * m2 - s1 * y - rou * m1 * s2) / ((powf(rou, 2) - 1) * powf(s2, 3) * s1);
                tmpVal =  -1.0 / (s2) + (m2 - y) * (-m2 * s1 + m2 * rou * s1 - rou * s2 * x + s1 * y) / ((powf(rou, 2) - 1) * powf(s2, 3) * s1);
                tmpPtr[thdIdx] += raw_y[batIdx * nodeNum + comb_dim + mixnodes] * raw_y[batIdx * nodeNum + comb_dim + mixIdx] * dyFeaMat[batIdx * comb_dim + nodeIdx]
                        * mix_surface[mixIdx * comb_dim + nodeIdx] * tmpVal;
            }
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++ incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        if (thdIdx == 0) {
            //dact[mixIdx * 6 + mixdimIdx] = 2.0 * sumVal / comb_dim * 10.0;
            dact[mixIdx * 6 + mixdimIdx] = sumVal * calc_exp(act[mixIdx * 6 + mixdimIdx]) + (act[mixIdx * 6 + mixdimIdx] - log(sqrtf(gridvar))) * mix_var_l2_penalty;
            //dact[mixIdx * 6 + mixdimIdx] = 0;
        }
    }
}

void ApplyGradMixVarsCUDA(NFloat* dact, NFloat* act, NFloat* dyFeaMat, NFloat* mix_surface, NFloat* raw_y, int batLen, int nodeNum, int comb_dim, int mixnodes, NFloat mix_var_l2_penalty, NFloat gridvar) {
    int blocks = mixnodes * 2;
    //int threads = batLen;
    int sbytes = THREADPERBLOCK * sizeof(float);
    int zblocks = CEIL(comb_dim, THREADPERBLOCK);
    //HKern_ResetZero<<< zblocks, THREADPERBLOCK >>>(dact, comb_dim);
    HKern_ApplyGradMixVars<<< blocks, THREADPERBLOCK, sbytes >>>(dact, act, dyFeaMat, mix_surface, raw_y, batLen, nodeNum, comb_dim, mixnodes, mix_var_l2_penalty, gridvar);
}




__global__ void HKern_ApplyGradMixMeans(NFloat* dact, NFloat* act, NFloat* dyFeaMat, NFloat* mix_surface, NFloat * raw_y, int batLen, int nodeNum, int comb_dim, int mixnodes, NFloat mix_mean_l2_penalty, NFloat* phonepos) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, batIdx, mixIdx, mixdimIdx, thdNum, dybase, mixbase, idx, incr, pos, nodeIdx;
    int total_jobs = batLen * comb_dim;
    NFloat sumVal, tmpVal, x, y, s1, s2, rou, m1, m2;
    int row_no, col_no;
    thdIdx = threadIdx.x;
    mixIdx = blockIdx.x / 2;
    mixdimIdx = blockIdx.x % 2;
    thdNum = min(blockDim.x, total_jobs);
    if (thdIdx < thdNum && mixIdx < mixnodes) { // && (mixIdx == 1 || mixIdx  == 10)) {
        idx = thdIdx;
        tmpPtr[thdIdx] = 0;

        m1 = act[mixIdx * 6 + 0];
        m2 = act[mixIdx * 6 + 1];
        s1 = calc_exp(act[mixIdx * 6 + 2]);
        rou= calc_tanh(act[mixIdx * 6 + 3]);
        s2 = calc_exp(act[mixIdx * 6 + 5]);
        //s1 = act[mixIdx * 6 + 2];
        //rou= act[mixIdx * 6 + 3];
        //s2 = act[mixIdx * 6 + 5];
        while (idx < total_jobs) {
            batIdx = idx / comb_dim;
            nodeIdx = idx % comb_dim;
            row_no = nodeIdx / 32;
            col_no = nodeIdx % 32;

            x = (0.5 + col_no) / 32.0;
            y = (0.5 + row_no) / 32.0;
            if (!mixdimIdx) {
                tmpVal = (m1 * s2 - m2 * rou * s1 + rou * s1 * y - s2 * x) / ((powf(rou, 2) - 1) * powf(s1, 2) * s2);
                                //scaler                                        * mixweights                                  * comb_dy
                tmpPtr[thdIdx] += raw_y[batIdx * nodeNum + comb_dim + mixnodes] * raw_y[batIdx * nodeNum + comb_dim + mixIdx] * dyFeaMat[batIdx * comb_dim + nodeIdx] 
                        //mix_dist
                        * mix_surface[mixIdx * comb_dim + nodeIdx] * tmpVal;// (((x - m1) / powf(s1, 2) - rou * (y - m2) / s1 / s2) / (1 - powf(rou, 2)));
            }
            else {
                tmpVal = (-m1 * rou * s2 + m2 * s1 + rou * s2 * x - s1 * y) / ((powf(rou, 2) - 1) * powf(s2, 2) * s1);
                tmpPtr[thdIdx] += raw_y[batIdx * nodeNum + comb_dim + mixnodes] * raw_y[batIdx * nodeNum + comb_dim + mixIdx] * dyFeaMat[batIdx * comb_dim + nodeIdx]
                        * mix_surface[mixIdx * comb_dim + nodeIdx] * tmpVal;//(((y - m2) / powf(s2, 2) - rou * (x - m1) / s1 / s2) / (1 - powf(rou, 2)));
            }
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++ incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        if (thdIdx == 0) {
            //dact[mixIdx * 6 + mixdimIdx] = 2.0 * sumVal / comb_dim * 10.0;
            dact[mixIdx * 6 + mixdimIdx] = sumVal + (act[mixIdx * 6 + mixdimIdx] - phonepos[mixIdx * 2 + mixdimIdx]) * mix_mean_l2_penalty;
            //dact[mixIdx * 6 + mixdimIdx] = 0;
        }
    }
}

void ApplyGradMixMeansCUDA(NFloat* dact, NFloat* act, NFloat* dyFeaMat, NFloat* mix_surface, NFloat* raw_y, int batLen, int nodeNum, int comb_dim, int mixnodes, NFloat mix_mean_l2_penalty, NFloat* phonepos) {
    int blocks = mixnodes * 2;
    //int threads = batLen;
    int sbytes = THREADPERBLOCK * sizeof(float);
    int zblocks = CEIL(comb_dim, THREADPERBLOCK);
    //HKern_ResetZero<<< zblocks, THREADPERBLOCK >>>(dact, comb_dim);
    HKern_ApplyGradMixMeans<<< blocks, THREADPERBLOCK, sbytes >>>(dact, act, dyFeaMat, mix_surface, raw_y, batLen, nodeNum, comb_dim, mixnodes, mix_mean_l2_penalty, phonepos);
}

__global__ void HKern_ScaleSil(NFloat *dest_y, NFloat *labMat, NFloat scaler, int scalerid, int batLen, int nodeNum) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < batLen * nodeNum) {
        int my_batid = idx / nodeNum;
        if (labMat[my_batid * nodeNum + scalerid] > 0.5) {
            dest_y[idx] = dest_y[idx] * scaler;
        }
    }
}
void ScaleSilCUDA(NFloat *dest_y, NFloat *labMat, NFloat scaler, int scalerid, int batLen, int nodeNum) {
    int nBlocks = CEIL(batLen * nodeNum, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ScaleSilCUDA: Block number exceeds the maximum");
    HKern_ScaleSil<<< nBlocks, THREADPERBLOCK >>>(dest_y, labMat, scaler, scalerid, batLen, nodeNum);
}

__global__ void HKern_ApplyDSigmoidActStimuMix(NFloat *srcMat, int len, NFloat *dstMat, int raw_dim, int comb_dim, int mixnodes, float resdnn_var, float mixscaler_var) {
    int pos;
    NFloat floatVal;
    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        int my_dim = pos % raw_dim;
        if (my_dim  < raw_dim - mixnodes - 1) {
            //floatVal = 1.0 - srcMat[pos] * srcMat[pos];
            //dstMat[pos] = floatVal / resdnn_var;
            //dstMat[pos] = (1.0 - srcMat[pos]) * srcMat[pos];
            dstMat[pos] = 1 - pow(srcMat[pos], 2);
        }
        else if (my_dim  >= raw_dim - mixnodes - 1 && my_dim < raw_dim - 1) {
            dstMat[pos] = (1.0 - srcMat[pos]) * srcMat[pos];
        }
        else {
            dstMat[pos] = (1.0 - srcMat[pos]) * srcMat[pos] / mixscaler_var;
            //dstMat[pos] = -1.0 * srcMat[pos];
            //dstMat[pos] = (1 - srcMat[pos]) * srcMat[pos] / comb_dim;
            //dstMat[pos] = -2.0 * srcMat[pos] * sqrt(-log(srcMat[pos]));
        }
    }
}

void ApplyDSigmoidActStimuMixCUDA(NFloat *srcMat, int len, NFloat *dstMat, int raw_dim, int comb_dim, float resdnn_var, float mixscaler_var) {
    int nBlocks = CEIL(len, THREADPERBLOCK);
    int mixnodes = raw_dim - comb_dim - 1;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDSigmoidActStimuMixCUDA: Block number exceeds the maximum");
    HKern_ApplyDSigmoidActStimuMix<<< nBlocks, THREADPERBLOCK >>>(srcMat, len, dstMat, raw_dim, comb_dim, mixnodes, resdnn_var, mixscaler_var);
}


__global__ void HKern_SplitGradDNNandMix_DNNpart(NFloat *dest_dy, NFloat *comb_dy, NFloat *comb_y, int batLen, int raw_dim, int comb_dim, float mixscaler, int tot) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tot) {
        return;
    }
    int col_id = idx % comb_dim;
    int bat_id = idx / comb_dim;
    //if (0.0 < comb_y[idx] && comb_y[idx] < 1.0) {
        dest_dy[bat_id * raw_dim + col_id] = mixscaler * comb_dy[idx];
    //}
    //else {
    //    dest_dy[bat_id * raw_dim + col_id] = 0;
    //}
}
__global__ void HKern_SplitGradDNNandMix_Mixpart(NFloat *dest_dy, NFloat *comb_dy, NFloat *raw_y, NFloat *mix_surface, int batLen, int raw_dim, int comb_dim, int mixnodes, float mixscaler) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, batIdx, mixIdx, thdNum, dybase, mixbase, idx, incr, pos;
    NFloat sumVal, tmpVal;
    thdIdx = threadIdx.x;
    batIdx = blockIdx.x / mixnodes;
    mixIdx = blockIdx.x % mixnodes;
    thdNum = min(blockDim.x, comb_dim);
    if (thdIdx < thdNum && batIdx < batLen && mixIdx < mixnodes) {
        dybase = batIdx * comb_dim;
        mixbase = mixIdx * comb_dim;
        idx = thdIdx;
        tmpPtr[thdIdx] = 0;
        while (idx < comb_dim) {
            tmpVal = comb_dy[dybase + idx] * mix_surface[mixbase + idx];
            tmpPtr[thdIdx] += tmpVal;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        if (thdIdx == 0) {
            dest_dy[batIdx * raw_dim + comb_dim + mixIdx] = sumVal * raw_y[batIdx * raw_dim + comb_dim + mixnodes]; // / mixnodes;
        }
    }
}
__global__ void HKern_SplitGradDNNandMix_Scalerpart(NFloat *dest_dy, NFloat *comb_dy, NFloat *raw_y, NFloat *comb_y, NFloat *mix_surface, int batLen, int raw_dim, int comb_dim, int mixnodes, float mixscaler) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, batIdx, mixIdx, thdNum, dybase, rawybase, idx, incr, pos;
    NFloat sumVal, tmpVal;
    thdIdx = threadIdx.x;
    batIdx = blockIdx.x;
    thdNum = min(blockDim.x, comb_dim);
    if (thdIdx < thdNum && batIdx < batLen) {
        dybase = batIdx * comb_dim;
        rawybase = batIdx * raw_dim;
        idx = thdIdx;
        tmpPtr[thdIdx] = 0;
        while (idx < comb_dim) {
            tmpVal = (comb_y[dybase + idx] - mixscaler * raw_y[rawybase + idx]) * comb_dy[dybase + idx];
            tmpPtr[thdIdx] += tmpVal;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        if (thdIdx == 0) {
            dest_dy[batIdx * raw_dim + comb_dim + mixnodes] = sumVal / raw_y[rawybase + comb_dim + mixnodes] / mixnodes; // / comb_dim / mixnodes;
        }
    }
}


__global__ void HKern_SplitGradDNNandMix_ZeroizeMixAndScaler(NFloat *dest_dy, int comb_dim, int raw_dim, int tot) {
    if (blockIdx.x * blockDim.x + threadIdx.x >= tot) {
        return;
    }
    int my_id = blockIdx.x * raw_dim + comb_dim + threadIdx.x;
    dest_dy[my_id] = 0.0;
}

void SplitGradDNNandMixCUDA(NFloat *dest_dy, NFloat *comb_dy, NFloat *raw_y, NFloat *comb_y, NFloat *mix_surface, int batLen, int raw_dim, int comb_dim, float mixscaler, Boolean enable_update_mix, Boolean enable_update_dnn) {
    //THREADPERBLOCK
    int tot = comb_dim * batLen;
    int nBlocks = CEIL(tot, THREADPERBLOCK);
    
    if (enable_update_dnn) {
        HKern_SplitGradDNNandMix_DNNpart<<< nBlocks, THREADPERBLOCK >>>(dest_dy, comb_dy, comb_y, batLen, raw_dim, comb_dim, mixscaler, tot);
    }
    else {
        HKern_SplitGradDNNandMix_DNNpart<<< nBlocks, THREADPERBLOCK >>>(dest_dy, comb_dy, comb_y, batLen, raw_dim, comb_dim, 0.0, tot);
    }
    
    if (!enable_update_mix) {
        int blkdim = raw_dim - comb_dim;
        nBlocks = batLen;
        HKern_SplitGradDNNandMix_ZeroizeMixAndScaler<<< nBlocks, blkdim >>>(dest_dy, comb_dim, raw_dim, blkdim * nBlocks);
        return;
    }
    else {
        int mixnodes = raw_dim - comb_dim - 1;
        nBlocks = mixnodes * batLen;
        int sBytes = THREADPERBLOCK * sizeof(NFloat);
        HKern_SplitGradDNNandMix_Mixpart<<< nBlocks, THREADPERBLOCK, sBytes >>>(dest_dy, comb_dy, raw_y, mix_surface, 
                batLen, raw_dim, comb_dim, mixnodes, mixscaler);
        //printf("comb_dim=%d,raw_dim=%d,mixnodes=%d\n", comb_dim, raw_dim, raw_dim - comb_dim - 1);
        nBlocks = batLen;
        HKern_SplitGradDNNandMix_Scalerpart<<< nBlocks, THREADPERBLOCK, sBytes >>>(dest_dy, comb_dy, raw_y, comb_y, mix_surface, 
                batLen, raw_dim, comb_dim, mixnodes, mixscaler);
    }
}

__global__ void HKern_ApplySigmoidActStimuMix(NFloat *srcPtr, int len, NFloat *dstPtr, int mixnodes, int raw_col, float resdnn_var, float mixscaler_var) {
    int pos;
    NFloat floatVal;
    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        int my_dim = pos % raw_col;
        if (my_dim  < raw_col - mixnodes - 1) {
            /*floatVal = 2.0 * srcPtr[pos] / resdnn_var;
            CHKNFLTEXPE(floatVal)
            floatVal = exp(floatVal);
            dstPtr[pos] = (floatVal - 1) / (floatVal + 1);
            */
            //floatVal = -1.0 * srcPtr[pos];
            //CHKNFLTEXPE(floatVal)
            //dstPtr[pos] = 1.0 / (1.0 + exp(floatVal));
            floatVal = srcPtr[pos];
            CHKNFLTEXPE(floatVal)
            floatVal = exp(floatVal);
            dstPtr[pos] = (floatVal - 1.0 / floatVal) / (floatVal + 1.0 / floatVal);
        }
        else if (my_dim >= raw_col - mixnodes - 1 && my_dim < raw_col - 1) {
            floatVal = srcPtr[pos];
            //CHKNFLTEXPE(floatVal)
            //dstPtr[pos] = exp(floatVal);
            dstPtr[pos] = floatVal;
        }
        else {
            //NFloat var = raw_col;
            //NFloat bias = log(mixnodes / sqrt(2.0 * PI * 0.1) + 70);
            //floatVal = -1.0 * srcPtr[pos] / var + bias;
            //CHKNFLTEXPE(floatVal)
            //dstPtr[pos] = exp(floatVal);
            //dstPtr[pos] = 1.0 / (1.0 + exp(floatVal));
            //if (dstPtr[pos] < 1e-7) {
            //    dstPtr[pos] = 1e-7;
            //}
            //floatVal = -1.0 * srcPtr[pos] * srcPtr[pos];
            //CHKNFLTEXPE(floatVal)
            //dstPtr[pos] = exp(floatVal);
            floatVal = -1.0 * srcPtr[pos] / mixscaler_var;
            CHKNFLTEXPE(floatVal)
            dstPtr[pos] = 1.0 / (1.0 + exp(floatVal));
            if (dstPtr[pos] < COMBLITTLE) {
                dstPtr[pos] = COMBLITTLE;
            }
        }
    }
}

void ApplySigmoidActStimuMixCUDA(NFloat *srcMat, int len, NFloat *dstMat, int mixnodes, int raw_col, float resdnn_var, float mixscaler_var) {
    int nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplySigmoidActStimuMixCUDA: Block number exceeds the maximum");
    //printf("mixscalervar=%f\n", mixscaler_var);
    HKern_ApplySigmoidActStimuMix<<<nBlocks, THREADPERBLOCK>>>(srcMat, len, dstMat, mixnodes, raw_col, resdnn_var, mixscaler_var);
}

__global__ void HKern_FwdCombDNNandMix(NFloat *comb_yFeaMat, NFloat *mix_surface, NFloat *raw_yFeaMat, int batLen, int raw_nodeNum, int comb_nodeNum, int mixnodes, float mixscaler) {
    extern __shared__ NFloat tmpPtr[];
    int my_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (blockIdx.x >= batLen * comb_nodeNum && threadIdx.x >= mixnodes) {
        return;
    }
    int one_bat_dim = comb_nodeNum * mixnodes;
    int my_batid = my_idx / one_bat_dim;
    int my_comb_col = (my_idx % one_bat_dim) / mixnodes;
    int my_mix_id = threadIdx.x;
    tmpPtr[my_mix_id] = mix_surface[my_mix_id * comb_nodeNum + my_comb_col] * raw_yFeaMat[my_batid * raw_nodeNum + comb_nodeNum + my_mix_id];
    __syncthreads();
    int thdIdx = threadIdx.x;
    int idx, incr, pos;
    for (idx = mixnodes; idx > 1; idx = incr) {
        incr = idx / 2;
        if (idx % 2 != 0) {
            ++incr;
        }
        if (thdIdx < incr) {
            pos = thdIdx + incr;
            if (pos < idx) {
                tmpPtr[thdIdx] += tmpPtr[pos];
            }
        }
        __syncthreads();
    }
    if (thdIdx == 0) {
        comb_yFeaMat[blockIdx.x] = /*raw_yFeaMat[my_batid * raw_nodeNum + comb_nodeNum + mixnodes] * tmpPtr[0] +*/ mixscaler * raw_yFeaMat[my_comb_col + my_batid * raw_nodeNum];
        //comb_yFeaMat[blockIdx.x] = tmpPtr[0] + mixscaler * raw_yFeaMat[my_comb_col + my_batid * raw_nodeNum];
        //comb_yFeaMat[blockIdx.x] = mixscaler * raw_yFeaMat[my_comb_col + my_batid * raw_nodeNum];
        /*if (comb_yFeaMat[blockIdx.x] < 0.0) {
            comb_yFeaMat[blockIdx.x] = 0.0;
        }
        if (comb_yFeaMat[blockIdx.x] > 1.0) {
            comb_yFeaMat[blockIdx.x] = 1.0;
        }
        */
        //if (comb_yFeaMat[blockIdx.x] < COMBLITTLE) {
        //    comb_yFeaMat[blockIdx.x] = COMBLITTLE;
        //}

    }
}

__global__ void HKern_CalcSoftmaxSum(NFloat *comb_softmaxSum, NFloat *raw_yFeaMat, int batLen, int raw_nodeNum, int comb_nodeNum, int mixnodes) {
    extern __shared__ NFloat tmpPtr[];
    int my_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (my_idx >= batLen * mixnodes) {
        return;
    }
    int thdIdx = threadIdx.x;
    int my_batid = blockIdx.x;
    tmpPtr[thdIdx] = raw_yFeaMat[my_batid * raw_nodeNum + comb_nodeNum + thdIdx];
    __syncthreads();
    int idx, incr, pos;
    for (idx = mixnodes; idx > 1; idx = incr) {
        incr = idx / 2;
        if (idx % 2 != 0) {
            ++incr;
        }
        if (thdIdx < incr) {
            pos = thdIdx + incr;
            if (pos < idx) {
                tmpPtr[thdIdx] += tmpPtr[pos];
            }
        }
        __syncthreads();
    }
    raw_yFeaMat[my_batid * raw_nodeNum + comb_nodeNum + thdIdx] /= tmpPtr[0];
    if (thdIdx == 0) {
        comb_softmaxSum[my_batid] = tmpPtr[0];
    }
}

__global__ void HKern_CalcSoftmaxMin(NFloat *comb_softmaxSum, NFloat *raw_yFeaMat, int batLen, int raw_nodeNum, int comb_nodeNum, int mixnodes) {
    extern __shared__ NFloat tmpPtr[];
    int my_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (my_idx >= batLen * mixnodes) {
        return;
    }
    int thdIdx = threadIdx.x;
    int my_batid = blockIdx.x;
    tmpPtr[thdIdx] = raw_yFeaMat[my_batid * raw_nodeNum + comb_nodeNum + thdIdx];
    __syncthreads();
    int idx, incr, pos;
    for (idx = mixnodes; idx > 1; idx = incr) {
        incr = idx / 2;
        if (idx % 2 != 0) {
            ++incr;
        }
        if (thdIdx < incr) {
            pos = thdIdx + incr;
            if (pos < idx) {
                if (tmpPtr[thdIdx] < tmpPtr[pos]) {
                    tmpPtr[thdIdx] = tmpPtr[pos];
                }
            }
        }
        __syncthreads();
    }
    raw_yFeaMat[my_batid * raw_nodeNum + comb_nodeNum + thdIdx] -= tmpPtr[0];
    
    NFloat tmpVal = raw_yFeaMat[my_batid * raw_nodeNum + comb_nodeNum + thdIdx];
    CHKNFLTEXPE(tmpVal)
    raw_yFeaMat[my_batid * raw_nodeNum + comb_nodeNum + thdIdx] = exp(tmpVal);
}


void FwdCombDNNandMixCUDA(NFloat *comb_yFeaMat, NFloat *mix_surface, NFloat *raw_yFeaMat, NFloat *comb_softmaxSum, int batLen, int raw_nodeNum, int comb_nodeNum, int mixnodes, float mixscaler) {
    int blks = comb_nodeNum * batLen;
    int sBytes = mixnodes * sizeof(NFloat);
    HKern_CalcSoftmaxMin<<< batLen, mixnodes, sBytes >>>(comb_softmaxSum, raw_yFeaMat, batLen, raw_nodeNum, comb_nodeNum, mixnodes);
    HKern_CalcSoftmaxSum<<< batLen, mixnodes, sBytes >>>(comb_softmaxSum, raw_yFeaMat, batLen, raw_nodeNum, comb_nodeNum, mixnodes);
    HKern_FwdCombDNNandMix<<< blks, mixnodes, sBytes >>>(comb_yFeaMat, mix_surface, raw_yFeaMat, batLen, raw_nodeNum, comb_nodeNum, mixnodes, mixscaler);
}

__global__ void HKern_ApplyActLHUCPenalty(NFloat *actscaler_elems, NFloat * dactscaler_elems, int nodeNum, float lhuc_penalty, int total_jobs, int grid_dim) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= total_jobs) {
        return;
    }
    int my_row = idx / grid_dim; 
    int my_col = idx % grid_dim;
    float tmpVal = 0;
    float myVal = actscaler_elems[idx];
    if (my_row - 1 >= 0) {
        tmpVal += myVal - actscaler_elems[(my_row - 1) * grid_dim + my_col];
    }
    if (my_row + 1 < grid_dim) {
        tmpVal += myVal - actscaler_elems[(my_row + 1) * grid_dim + my_col];
    }
    if (my_col - 1 >= 0) {
        tmpVal += myVal - actscaler_elems[idx - 1];
    }
    if (my_col + 1 < grid_dim) {
        tmpVal += myVal - actscaler_elems[idx + 1];
    }
    dactscaler_elems[idx] += lhuc_penalty * tmpVal;
}

void ApplyActLHUCPenaltyCUDA(NFloat * actscaler_elems, NFloat * dactscaler_elems, int nodeNum, float lhuc_penalty) {
    int total_jobs = nodeNum;
    int nBlocks = CEIL(total_jobs, THREADPERBLOCK);
    int grid_dim = (int)sqrt(nodeNum);
    HKern_ApplyActLHUCPenalty<<<nBlocks, THREADPERBLOCK>>>(actscaler_elems, dactscaler_elems, nodeNum, lhuc_penalty, total_jobs, grid_dim);
}





__global__ void HKern_ApplyActLHUCPenaltySoftCUDA(NFloat * d_lhuc, NFloat * lhuc, NFloat * phoneidx, NFloat * pos_dist, float lhuc_range_var, float lhuc_dist_var, int nodeNum, int batLen, NFloat * penal_val) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx = threadIdx.x; 
    int blkIdx = blockIdx.x;
    int thdNum = min(blockDim.x, nodeNum);
    
    int idx, base, pos, pos_dist_base, my_node_id, incr;

    float regVal = 0;
    float tmpVal, my_pos_dist, idx_pos_dist, my_lhuc, imp2, deltaVal;
    if (thdIdx < thdNum && blkIdx < batLen * nodeNum) {
        pos_dist_base = ((int)phoneidx[blkIdx / nodeNum]) * nodeNum;
        my_node_id = blkIdx % nodeNum;
        idx = thdIdx;
        tmpPtr[thdIdx] = 0;
        my_pos_dist = pos_dist[pos_dist_base + my_node_id];
        imp2 = exp(-my_pos_dist * my_pos_dist / lhuc_dist_var);
        my_lhuc = lhuc[my_node_id];
        while (idx < nodeNum) {
            idx_pos_dist = pos_dist[pos_dist_base + idx];
            tmpVal = idx_pos_dist - my_pos_dist;
            tmpVal = exp(-tmpVal * tmpVal / lhuc_range_var);
            regVal += tmpVal * (my_lhuc - lhuc[idx]) * (my_lhuc - lhuc[idx]) * imp2;
            deltaVal = tmpVal * (my_lhuc - lhuc[idx]) * imp2 + 
                    exp(-idx_pos_dist * idx_pos_dist / lhuc_dist_var) * tmpVal * (my_lhuc - lhuc[idx]);
            tmpPtr[thdIdx] += deltaVal;
            idx += thdNum;
        }
        atomicAdd(penal_val, regVal);
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        
        if (thdIdx == 0) {
            d_lhuc[blkIdx] = 2 * tmpPtr[0]; // * exp(-my_pos_dist * my_pos_dist / lhuc_dist_var);
        }
    }
}


__global__ void HKern_ApplyActLHUCPenaltySoftAddCUDA(NFloat * d_lhuc, NFloat * dest_d_lhuc, int nodeNum, int batLen, float lhuc_penalty) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx = threadIdx.x;
    int blkIdx = blockIdx.x;
    int thdNum = min(blockDim.x, batLen);
    int idx, pos, my_node_id, incr, step, tot;
    if (thdIdx < thdNum && blkIdx < nodeNum) {
        
        idx = thdIdx;
        step = thdNum * nodeNum;
        tmpPtr[thdIdx] = 0;
        tot = nodeNum * batLen;
        while (idx < tot) {
            tmpPtr[thdIdx] += d_lhuc[idx];
            idx += step;
        }
        __syncthreads();
        
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++ incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        if (thdIdx == 0) {
            /*
            int j;
            for (j = 0; j < thdNum; j += 1) {
                tmpPtr[0] += d_lhuc[j];
            }
            */
            dest_d_lhuc[blkIdx] += lhuc_penalty * tmpPtr[0];
        }
    }
}

void ApplyActLHUCPenaltySoftCUDA(NFloat * d_lhuc, NFloat * lhuc, NFloat * dest_d_lhuc, NFloat * phoneidx, NFloat * pos_dist, float lhuc_range_var, float lhuc_dist_var, int nodeNum, int batLen, float lhuc_penalty, NFloat * penal_val) {
    int nBlocks = batLen * nodeNum;
    int sBytes = THREADPERBLOCK * sizeof(NFloat);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyActLHUCPenaltySoftCUDA: Block number exceeds the maximum");
    HKern_ApplyActLHUCPenaltySoftCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(d_lhuc, lhuc, phoneidx, pos_dist, lhuc_range_var, lhuc_dist_var, nodeNum, batLen, penal_val);
    //HKern_ApplyActLHUCPenaltyL2CUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(d_lhuc, lhuc, phoneidx, pos_dist, lhuc_range_var, lhuc_dist_var, nodeNum, batLen, penal_val);
    nBlocks = nodeNum;
    HKern_ApplyActLHUCPenaltySoftAddCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(d_lhuc, dest_d_lhuc, nodeNum, batLen, lhuc_penalty);
}


__global__ void HKern_ApplyActLHUCPenaltyLocalSoftCUDA(NFloat * lhuc, NFloat * dest_d_lhuc, NFloat * d_lhuc_surface, int nodeNum, int batLen, float lhuc_penalty, NFloat * penalty_val) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx = threadIdx.x; 
    int blkIdx = blockIdx.x;
    int thdNum = min(blockDim.x, nodeNum);
    int idx, base, pos, incr;

    float regVal = 0;
    float tmpVal, my_lhuc;
    if (thdIdx < thdNum && blkIdx < nodeNum) {
        my_lhuc = lhuc[blkIdx];
        idx = thdIdx;
        tmpPtr[thdIdx] = 0;
        d_lhuc_surface += blkIdx * nodeNum + idx;
        while (idx < nodeNum) {
            tmpVal = my_lhuc - lhuc[idx];
            tmpPtr[thdIdx] += tmpVal * (*d_lhuc_surface);
            regVal += tmpVal * tmpVal * (*d_lhuc_surface);
            idx += thdNum;
            d_lhuc_surface += thdNum;
        }
        atomicAdd(penalty_val, 4 * regVal);
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        if (thdIdx == 0) {
            dest_d_lhuc[blkIdx] += lhuc_penalty * tmpPtr[0];
        }
    }
}
void ApplyActLHUCPenaltyLocalSoftCUDA(NFloat * lhuc, NFloat * dest_d_lhuc, NFloat * d_lhuc_surface, int nodeNum, int batLen, float lhuc_penalty, NFloat * penalty_val) {
    int nBlocks = nodeNum;
    int sBytes = THREADPERBLOCK * sizeof(NFloat);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyActLHUCPenaltyLocalSoftCUDA: Block number exceeds the maximum");
    //printf("ksksksk=%f\n", lhuc_penalty);exit(0);
    HKern_ApplyActLHUCPenaltyLocalSoftCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(lhuc, dest_d_lhuc, d_lhuc_surface, nodeNum, batLen, lhuc_penalty, penalty_val);
}


__global__ void HKern_StimuGrad(NFloat * srcPtr, int row, int col, NFloat * grid_suface, int num_phone, NFloat * phone_vec, NFloat *dstPtr, NFloat * weight_norms) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos;
    NFloat sumVal, tmpVal;
    NFloat a;
    

    thdIdx = threadIdx.x;   /* num threads per block */
    rowIdx = blockIdx.x;    /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);


    if (thdIdx < thdNum && rowIdx < row) {
        /* 1. find the sum Z */
        /* a. collect the sum for the groups */
        base = rowIdx * col;

        idx = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (idx < col) {
            pos = base + idx;
            dstPtr[pos] = srcPtr[pos];
            if (dstPtr[pos] < LITTLE) {
                dstPtr[pos] = dstPtr[pos];
            }
            dstPtr[pos] *= weight_norms[idx];
            tmpPtr[thdIdx] += dstPtr[pos];
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        sumVal = tmpPtr[0];

        /* 3. normalise */
        idx = thdIdx; 
        while (idx < col) {
            //if (abs(dstPtr[base + idx]) < LITTLE) {
            //    dstPtr[base + idx] = LITTLE;
            //}
            a = (1 / sumVal - grid_suface[(idx) * num_phone + (int)(phone_vec[rowIdx])] / dstPtr[base + idx]) * weight_norms[idx];
            dstPtr[base + idx] = a;
            idx += thdNum;
        }
    }
}

void StimuGradCUDA(NFloat * srcPtr, int row, int col, NFloat * grid_suface, int num_phone, NFloat * phone_vec, NFloat *dstPtr, NFloat * weight_norms) {
    int nBlocks, sBytes;

    //HError(9999, "row=%d col=%d num_phone=%d tpb=%d\n", row, col, num_phone, THREADPERBLOCK);

    nBlocks = row;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "StimuGradCUDA: Block number exceeds the maximum");
    HKern_StimuGrad<<<nBlocks, THREADPERBLOCK, sBytes>>>(srcPtr, row, col, grid_suface, num_phone, phone_vec, dstPtr, weight_norms);
}


__global__ void HKern_CalcWeightNorms(NFloat * weight_norms, NFloat * weights, int input_dim, int output_dim) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos, step, total_weight;
    NFloat sumVal;


    thdIdx = threadIdx.x;
    rowIdx = blockIdx.x;
    thdNum = min(blockDim.x, output_dim);

    step = thdNum * input_dim;
    total_weight = input_dim * output_dim;

    if (thdIdx < thdNum && rowIdx < input_dim) {
        /*if (thdIdx == 0) {
            int xx = 0;
            weight_norms[rowIdx] = 0;
            for (xx = 0; xx < output_dim; ++ xx) {
                weight_norms[rowIdx] += weights[xx * input_dim + rowIdx] * weights[xx * input_dim + rowIdx];
            }
        }
        */
        
        base = thdIdx * input_dim + rowIdx;
        tmpPtr[thdIdx] = weights[base] * weights[base];
        base += step;
        while (base < total_weight) {
            tmpPtr[thdIdx] += weights[base] * weights[base];
            base += step;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        if (thdIdx == 0) {
            weight_norms[rowIdx] = sqrt(sumVal);
        }
    }
}

void CalcWeightNormsCUDA(NFloat * weight_norms, NFloat * weights, int input_dim, int output_dim) {
    int nBlocks, sBytes;
    nBlocks = input_dim;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "CalcWeightNormsCUDA: Block number exceeds the maximum");
    HKern_CalcWeightNorms<<<nBlocks, THREADPERBLOCK, sBytes>>>(weight_norms, weights, input_dim, output_dim);
}

__global__ void HKern_ApplyActStimuPenalty(NFloat * stimu_dy, NFloat * y, NFloat * actscaler, NFloat * dactscaler, int batLen, int nodeNum, float stimu_penalty) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos, step, total_weight;
    NFloat sumVal;
    thdIdx = threadIdx.x;
    rowIdx = blockIdx.x;
    thdNum = min(blockDim.x, batLen);

    step = thdNum * nodeNum;
    total_weight = batLen * nodeNum;
    if (thdIdx < thdNum && rowIdx < nodeNum) {
        base = thdIdx * nodeNum + rowIdx;
        tmpPtr[thdIdx] = stimu_dy[base] * y[base];
        base += step;
        while (base < total_weight) {
            tmpPtr[thdIdx] += stimu_dy[base] * y[base];
            base += step;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        if (thdIdx == 0) {
            dactscaler[rowIdx] += stimu_penalty * sumVal / actscaler[rowIdx];
            //dactscaler[rowIdx] = -100;
        }
    }
}

void ApplyActStimuPenaltyCUDA(NFloat * stimu_dy, NFloat * y, NFloat * actscaler, NFloat * dactscaler, int batLen, int nodeNum, float stimu_penalty) {
    int nBlocks = nodeNum;
    int sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyActStimuPenaltyCUDA: Block number exceeds the maximum");
    HKern_ApplyActStimuPenalty<<<nBlocks, THREADPERBLOCK, sBytes>>>(stimu_dy, y, actscaler, dactscaler, batLen, nodeNum, stimu_penalty);
}

__global__ void HKern_CalcSumActi(NFloat * sum_xFeaMat, NFloat * xFeaMats, NFloat * weight_norms, int batLen, int nodeNum) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos;
    NFloat sumVal, tmpVal;

    thdIdx = threadIdx.x;
    rowIdx = blockIdx.x;
    thdNum = min(blockDim.x, nodeNum);

    if (thdIdx < thdNum && rowIdx < batLen) {
        base = rowIdx * nodeNum;
        idx = thdIdx;
        /*if (idx < nodeNum) {
            tmpVal = xFeaMats[base + idx];
            if (tmpVal < LITTLE) tmpVal = LITTLE;
            tmpPtr[thdIdx] = tmpVal *  weight_norms[idx];
            idx += thdNum;
            if (idx < nodeNum) {
                tmpVal = xFeaMats[base + idx];
                tmpPtr[thdIdx] = tmpVal *  weight_norms[idx];
            }
        }
        */
        tmpPtr[thdIdx] = 0;
        while (idx < nodeNum) {
            tmpVal = xFeaMats[base + idx] * weight_norms[idx];
            if (tmpVal < LITTLE) tmpVal = LITTLE;
            tmpPtr[thdIdx] += tmpVal;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        
        if (thdIdx == 0) {
            sum_xFeaMat[rowIdx] = sumVal;
        }
        
        /*if (thdIdx == 0) {
            int j;
            sumVal = 0;
            for (j = 0; j < nodeNum; ++ j) {
               sumVal += xFeaMats[base + j] * weight_norms[j];
            }
            sum_xFeaMat[rowIdx] = sumVal;
        }
        */
    }
}

void CalcSumActiCUDA(NFloat * sum_xFeaMat_elems, NFloat * xFeaMats, NFloat * weight_norms, int batLen, int nodeNum) {
    int nBlocks, sBytes;
    nBlocks = batLen;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "CalcSumActiCUDA: Block number exceeds the maximum");
    HKern_CalcSumActi<<<nBlocks, THREADPERBLOCK, sBytes>>>(sum_xFeaMat_elems, xFeaMats, weight_norms, batLen, nodeNum);
}

__global__ void HKern_CalcStimuKL(NFloat *klvec, NFloat *yFeaMats, NFloat *weight_norms, NFloat *sum_yFeaMats, NFloat *acti_surface, NFloat *phoneidx_vec, int batLen, int nodeNum, int num_phone) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, thdNum, idx, incr, pos;
    NFloat sumVal, surfVal, tmpVal;

    thdIdx = threadIdx.x;
    int total = batLen * nodeNum;
    thdNum = min(blockDim.x, total);



    //int num_phone = 46; //TODO
    int phone_id, batid, nodeid;

    
    


    if (thdIdx < thdNum) {
        
        idx = thdIdx;

        
        tmpPtr[thdIdx] = 0;
        //tmpPtr[thdIdx] = surfVal * log(surfVal / (surfVal + 0.1));

        while (idx < total) {
            batid = idx / nodeNum;
            nodeid = idx % nodeNum;
            phone_id = (int)(phoneidx_vec[batid]);
            surfVal = acti_surface[phone_id + num_phone * nodeid];
            tmpVal = yFeaMats[idx];
            if (tmpVal < LITTLE) {
                tmpVal = LITTLE;
            }
            tmpPtr[thdIdx] += surfVal * log(surfVal * sum_yFeaMats[batid] / tmpVal / weight_norms[nodeid]);
            //tmpPtr[thdIdx] += surfVal * log(surfVal / (surfVal + 0.1));
            
            idx += thdNum;
            
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        sumVal = tmpPtr[0];
        if (thdIdx == 0) {
            klvec[0] = sumVal;
        }
    }
}

void CalcStimuKLCUDA(NFloat *klvec, NFloat *yFeaMats, NFloat *weight_norms, NFloat *sum_yFeaMats, NFloat *acti_surface, NFloat *phoneidx_vec, int batLen, int nodeNum, int num_phone) {
    int sBytes = sizeof(NFloat) * THREADPERBLOCK;
    HKern_CalcStimuKL<<<1, THREADPERBLOCK, sBytes>>>(klvec, yFeaMats, weight_norms, sum_yFeaMats, acti_surface, phoneidx_vec, batLen, nodeNum, num_phone);
}



//cw564 - stimu -- end


__global__ void HKern_ApplyRedSoftmaxAct(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos;
    NFloat maxVal, sumVal, tmpVal;

    thdIdx = threadIdx.x;	/* num threads per block */
    rowIdx = blockIdx.x;	/* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);
    if (thdIdx < thdNum && rowIdx < row) {
        base = rowIdx * col;
        /* 1. find the max val for current frame (rowIdx) and store it in tmpPtr[thdIdx] */
        /* a. collect the maxes for the groups */
        idx = thdIdx;
        tmpPtr[thdIdx] = srcPtr[base + idx];
        idx += thdNum;
        while (idx < col) {
            pos = base + idx;
            if (tmpPtr[thdIdx] < srcPtr[pos])
                tmpPtr[thdIdx] = srcPtr[pos];
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual max within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx && tmpPtr[thdIdx] < tmpPtr[pos]) {
                    tmpPtr[thdIdx] = tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        maxVal = tmpPtr[0];
        __syncthreads();
        /* 2. find the sum */
        /* a. collect the sum for the groups */
        idx = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (idx < col) {
            pos = base + idx;
            tmpVal = srcPtr[pos] - maxVal;
            CHKNFLTEXPE(tmpVal)
            dstPtr[pos] = exp(tmpVal);
            tmpPtr[thdIdx] += dstPtr[pos];
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        sumVal = tmpPtr[0];
        /* 3. normalise */
        idx = thdIdx; 
        while (idx < col) {
            dstPtr[base + idx] /= sumVal;
            idx += thdNum;
        }
    } 
}

__global__ void HKern_ApplySoftReLAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = log(1.0 + exp(floatVal));
    } 
}

__global__ void HKern_ApplyDSoftReLAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        CHKNFLTEXPE(floatVal)
        dstPtr[pos] = 1.0 - 1.0 / exp(floatVal);
    }
}

__global__ void HKern_ApplySoftSignAct(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        dstPtr[pos] = srcPtr[pos] / (1 + abs(srcPtr[pos]));
    }
}

__global__ void HKern_ApplyLogTrans(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int pos;
    NFloat floatVal;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        floatVal = srcPtr[pos];
        if (floatVal <= 0) {
            floatVal = LZERO;
        }
        else {        
            floatVal = log(floatVal);
            if (floatVal < LSMALL) {
                floatVal = LSMALL;
            }
        }
        dstPtr[pos] = floatVal;
    }
}

__global__ void HKern_RedSumNMatrixByColCUDA(NFloat *srcPtr, int row, int col, Boolean accFlag, NFloat *dstPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, colIdx, thdNum, base, idx, incr, pos, step;

    thdIdx = threadIdx.x;       /* num threads per block */
    colIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, row);
    if (thdIdx < thdNum && colIdx < col) {
        /*base = colIdx;*/
        /* collect the sums for the groups (and transpose the matrix) */
        tmpPtr[thdIdx] = 0.0;/*srcPtr[base + idx * col];*/
        base = colIdx;
        idx = thdIdx;
        pos = base + idx * col;
        step = thdNum * col;
        while (idx < row) {
            tmpPtr[thdIdx] += srcPtr[pos];
            pos += step;
            idx += thdNum;
        }
        __syncthreads();
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }

        /* copy the value to dstPtr */
        if (thdIdx == 0) {
            if (accFlag == FALSE) 
                dstPtr[colIdx] = 0.0; 
            dstPtr[colIdx] += tmpPtr[0];
        }
    }
}

__global__ void HKern_SumNMatrixByCol(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int i, pos;
    NFloat sum;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < col) {
        sum = 0.0;
        for (i = 0; i < row; ++i) {
            sum += srcPtr[i * col + pos];
        }
        dstPtr[pos] = sum;
    }
}

__global__ void HKern_SumNMatrixByColAcc(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int i, pos;
    NFloat sum;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < col) {
        sum = 0.0;
        for (i = 0; i < row; ++i) {
            sum += srcPtr[i * col + pos];
        }
        dstPtr[pos] += sum;
    }
}

__global__ void HKern_SquaredNSegment(NFloat *srcPtr, int segLen, NFloat *dstPtr) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        dstPtr[pos] = pow(srcPtr[pos], 2);
    }
}

__global__ void HKern_CompAdaGradNSegment(NFloat eta, int K, int segLen, NFloat *ssgSeg, NFloat *nlrSeg) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        nlrSeg[pos] = eta / sqrt(K + ssgSeg[pos]);
    }
}

__global__ void HKern_CalXENTCriterionCUDA(NFloat *refPtr, NFloat *hypPtr, int segLen, NFloat *crtPtr) {
    __shared__ NFloat tmpPtr[THREADPERBLOCK];
    int thdIdx, thdNum, pos, idx, incr;
    NFloat tn, yn;

    thdIdx = threadIdx.x;
    thdNum = blockDim.x;

    if (thdIdx < thdNum) {
        /* a. collect the sums for the groups */
        pos = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (pos < segLen) {
            tn = refPtr[pos];
            yn = hypPtr[pos];
            if (tn == 0.0) {
                tmpPtr[thdIdx] += 0.0;
            }
            else if (yn == 0.0) {
                tmpPtr[thdIdx] += tn * LZERO;
            }
            else {
                tmpPtr[thdIdx] += (-1.0) * tn * log(yn / tn); 
            }
            pos += thdNum;
        }
        __syncthreads();
        /* b. dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        *crtPtr = tmpPtr[0];
    } 
}

__global__ void HKern_CalMMSECriterionCUDA(NFloat *refPtr, NFloat *hypPtr, int segLen, NFloat *crtPtr) {
    __shared__ NFloat tmpPtr[THREADPERBLOCK];
    int thdIdx, thdNum, pos, idx, incr;

    thdIdx = threadIdx.x;
    thdNum = blockDim.x;
    
    if (thdIdx < thdNum) {
        /* a. collect the sums for the groups */
        pos = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (pos < segLen) {
            tmpPtr[thdIdx] += pow(refPtr[pos] - hypPtr[pos], 2);
            pos += thdNum;
        }
        __syncthreads();
        /* dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        *crtPtr = tmpPtr[0];
    }
}

__global__ void HKern_AddSegmentTargetPen(NFloat *srcPtr, NFloat *penPtr, int row, int col, NFloat *dstPtr) {
    int pos, off;
    
    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < row * col) {
        off = pos % col;
        dstPtr[pos] = srcPtr[pos] + penPtr[off];
    }
}

/*__global__ void HKern_SubNSegmentByConst(NFloat *srcSeg, int segLen, float constVal, NFloat *dstSeg) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        dstSeg[pos] = srcSeg[pos] - constVal;
    }
}*/

/* cz277 - semi */
__global__ void HKern_ShiftNSegmentVals(NFloat *srcSeg, int segLen, float shiftVal, NFloat *dstSeg) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < segLen) {
        dstSeg[pos] = srcSeg[pos] + shiftVal;
    }
}

/* cz277 - 1007 */
__global__ void HKern_CopyPartialNSegment(int minRow, int minCol, NFloat *srcPtr, int srcCol, NFloat *dstPtr, int dstCol) {
    int pos, rowIdx, colIdx;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < minRow * minCol) {
        rowIdx = pos / minCol;
        colIdx = pos % minCol;
        dstPtr[rowIdx * dstCol + colIdx] = srcPtr[rowIdx * srcCol + colIdx];
    }
}

/* cz277 - gradlim */
__global__ void HKern_ClipNSegmentVals(NFloat* srcSeg, int len, NFloat upperLim, NFloat lowerLim, NFloat *dstSeg) {
    int pos;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < len) {
        if (srcSeg[pos] > upperLim)
            dstSeg[pos] = upperLim;
        else if (srcSeg[pos] < lowerLim)
            dstSeg[pos] = lowerLim;
        else if (srcSeg != dstSeg)
            dstSeg[pos] = srcSeg[pos];
    }
}

__global__ void HKern_RedMaxElementIndex(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos, off = THREADPERBLOCK;

    thdIdx = threadIdx.x;       /* num threads per block */
    rowIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);
    if (thdIdx < thdNum && rowIdx < row) {
        base = rowIdx * col;
        /* find the max val for current frame (rowIdx) and store it in tmpPtr[thdIdx] */
        /* a. collect the maxes for the groups */
        idx = thdIdx;
        tmpPtr[thdIdx] = srcPtr[base + idx];
        tmpPtr[off + thdIdx] = idx;
        idx += thdNum;
        while (idx < col) {
            pos = base + idx;
            if (tmpPtr[thdIdx] < srcPtr[pos]) {
                tmpPtr[thdIdx] = srcPtr[pos];
                tmpPtr[off + thdIdx] = idx;
            }
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual max within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx && tmpPtr[thdIdx] < tmpPtr[pos]) {
                    tmpPtr[thdIdx] = tmpPtr[pos];
                    tmpPtr[off + thdIdx] = tmpPtr[off + pos];
                }
            }
            __syncthreads();
        }
        /*__syncthreads();*/
        if (thdIdx == 0)
            dstPtr[rowIdx] = tmpPtr[off + 0];
            /*dstPtr[rowIdx] = (NFloat) tmpPtr[off + 0];*/
        /*__syncthreads();*/
    }	
}

/* cz277 - max norm */
__global__ void HKern_RedCalExtNMatrixL2Norm(NFloat *matPtr, NFloat *vecPtr, int row, int col, NFloat *alphas) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos;

    thdIdx = threadIdx.x;       /* num threads per block */
    rowIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);
    if (thdIdx < thdNum && rowIdx < row) {
        /* 1. accumulate the L2 norm for each row */
        base = rowIdx * col;
        idx = thdIdx;
        tmpPtr[thdIdx] = 0.0;
        while (idx < col) {
            pos = base + idx;
            tmpPtr[thdIdx] += pow(matPtr[pos], 2);
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual add within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;                                   
	    }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx) {
                    tmpPtr[thdIdx] += tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        
        if (thdIdx == 0)
            alphas[rowIdx] = tmpPtr[0] + pow(vecPtr[rowIdx], 2);
    }
}

/* cz277 - max norm */
__global__ void HKern_RedMaxElementValue(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    extern __shared__ NFloat tmpPtr[];
    int thdIdx, rowIdx, thdNum, base, idx, incr, pos;

    thdIdx = threadIdx.x;       /* num threads per block */
    rowIdx = blockIdx.x;        /* block index, i.e., row num (minibatch size)  */
    thdNum = min(blockDim.x, col);
    if (thdIdx < thdNum && rowIdx < row) {
        base = rowIdx * col;
        /* find the max val for current frame (rowIdx) and store it in tmpPtr[thdIdx] */
        /* a. collect the maxes for the groups */
        idx = thdIdx;
        tmpPtr[thdIdx] = srcPtr[base + idx];
        idx += thdNum;
        while (idx < col) {
            pos = base + idx;
            if (tmpPtr[thdIdx] < srcPtr[pos]) {
                tmpPtr[thdIdx] = srcPtr[pos];
            }
            idx += thdNum;
        }
        __syncthreads();
        /* b. dual max within current block */
        for (idx = thdNum; idx > 1; idx = incr) {
            incr = idx / 2;
            if (idx % 2 != 0) {
                ++incr;
            }
            if (thdIdx < incr) {
                pos = thdIdx + incr;
                if (pos < idx && tmpPtr[thdIdx] < tmpPtr[pos]) {
                    tmpPtr[thdIdx] = tmpPtr[pos];
                }
            }
            __syncthreads();
        }
        /*__syncthreads();*/
        if (thdIdx == 0)
            dstPtr[rowIdx] = tmpPtr[0];
    }
}

/* --------------------------- HFBLat Kerns ------------------------ */

/* cz277 - cuda fblat */
__global__ void HKern_Setotprob4q(int T, NFloat *llhPtr, int ncols, int *qLo, int *qHi, int Q, float probScale, AcousticDev *acList) {
    int pos, tIdx, tRel, qIdx, s, Nq1;
    AcousticDev *curAc;
    NFloat *otprob;
    NFloat *matptr;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < T * Q) {
        tIdx = pos / Q + 1;
        qIdx = pos % Q + 1;
        if (qIdx >= qLo[tIdx] && qIdx <= qHi[tIdx]) {
            curAc = &acList[qIdx];
            Nq1 = curAc->Nq + 1;
            if (tIdx >= curAc->t_start && tIdx <= curAc->t_end) {	/* q is active at t */
                matptr = llhPtr + (tIdx - 1) * ncols;
                tRel = tIdx - curAc->t_start + 1;
                otprob = curAc->otprob + tRel * Nq1;
                for (s = 2; s < curAc->Nq; ++s) {
                    otprob[s] = matptr[curAc->indexes[s] - 1];
                }
            }
        }
    }
}


/* cz277 - cuda fblat */
__device__ NFloat LAddDev(NFloat x, NFloat y) {
    NFloat temp, diff, z;

    if (x < y) {
        temp = x;
        x = y;
        y = temp;
    }
    diff = y - x;
    if (diff < -23.025851) {
        if (x < LSMALL) {
            return LZERO;
        }
        else {
            return x;
        }
    }
    else {
        z = exp(diff);
        return x + log(1.0 + z);
    }
}

/* cz277 - cuda fblat */
__global__ void HKern_SetModelPlus(int Q, AcousticDev *acList) {
    int tIdx, tRel, qIdx, Nq1, i, j;
    AcousticDev *curAc;
    NFloat *bqt, *bqt1, x;

    qIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (qIdx < Q) {
        qIdx += 1;
        curAc = acList + qIdx;
        Nq1 = curAc->Nq + 1;
        for (tIdx = curAc->t_end; tIdx >= curAc->t_start; --tIdx) {
            tRel = tIdx - curAc->t_start + 1;
            /* SetModelPlus subroutine */
            x = LZERO;
            bqt = &curAc->betaPlus[tRel * Nq1];
            bqt1 = &curAc->betaPlus[(tRel + 1) * Nq1];
            if (tIdx == curAc->t_end) 
                bqt[curAc->Nq] = 0;
            else 
                bqt[curAc->Nq] = LZERO;
            for (i = 2; i < curAc->Nq; ++i) {
                x = bqt[curAc->Nq] + curAc->transp[i * Nq1 + curAc->Nq]; 
                if (tIdx + 1 <= curAc->t_end) {	/* in beam next time frame */
                    for (j = 2; j < curAc->Nq; ++j) {
                        x = LAddDev(x, bqt1[j] + curAc->transp[i * Nq1 + j]);
                    }
                }
                x += curAc->otprob[tRel * Nq1 + i];
                bqt[i] = x;
            }
            x = LZERO;
            for (i = 2; i < curAc->Nq; ++i) {
                x = LAddDev(x, bqt[i] + curAc->transp[1 * Nq1 + i]);
            }
            bqt[1] = x;
        }
        /* neet to set the total accumulated acoustics (tRel ~ tIdx = curAc->t_start) */
        if (curAc->SP == TRUE)
            curAc->aclike = curAc->transp[1 * Nq1 + curAc->Nq];
        else
            curAc->aclike = curAc->betaPlus[tRel * Nq1 + 1];
    }
}


/* cz277 - cuda fblat */
__global__ void HKern_ZeroAlphas(int T, int Q, AcousticDev *acList) {
    int i, pos, Nq1, tIdx, tRel, qIdx;
    AcousticDev *curAc;
    NFloat *alpha;

    pos = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (pos < T * Q) {
        tIdx = pos / Q + 1;
        qIdx = pos % Q + 1;
        curAc = &acList[qIdx];
        /* q is active at t */
        if (tIdx >= curAc->t_start && tIdx <= curAc->t_end) { 
            tRel = tIdx - curAc->t_start + 1;
            Nq1 = curAc->Nq + 1;
            alpha = &curAc->alphaPlus[tRel * Nq1];
            if (curAc->SP == FALSE) {
                for (i = 1; i < Nq1; ++i) {
                    alpha[i] = LZERO;    
                }
            }
        }
    }
}


/* cz277 - cuda fblat */
__global__ void HKern_StepAlpha(int Q, AcousticDev *acList) {
    int tIdx, qIdx, Nq1, i, j, tRel;
    AcousticDev *curAc;
    NFloat *aq, *laq, x = 0.0, y, a;
    NFloat *outprob;

    qIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (qIdx < Q) {
        qIdx += 1;
        curAc = acList + qIdx;
        /* for each time */
        for (tIdx = curAc->t_start; tIdx <= curAc->t_end; ++tIdx) {
            tRel = tIdx - curAc->t_start + 1;
            Nq1 = curAc->Nq + 1;
            aq = &curAc->alphaPlus[tRel * Nq1];
            laq = (tIdx - 1 >= curAc->t_start && tIdx - 1 <= curAc->t_end)? &curAc->alphaPlus[(tRel - 1) * Nq1]: NULL;
            /* outprob != NULL ?? */
            outprob = &curAc->otprob[tRel * Nq1];
            if (tIdx == curAc->t_start) 
                aq[1] = curAc->locc - curAc->aclike;
            else 
                aq[1] = LZERO;
            x = LZERO;
            for (j = 2; j < curAc->Nq; ++j) {
                a = curAc->transp[1 * Nq1 + j];
                x = (a > LSMALL)? a + aq[1]: LZERO;
                for (i = 2; i <= curAc->Nq; ++i) {
                    a = curAc->transp[i * Nq1 + j];
                    y = (laq? laq[i]: LZERO);
                    if (a > LSMALL && y > LSMALL) {
                        x = LAddDev(x, y + a);
                        /*x = log(x + y + a);*/
                    }
                }
                aq[j] = x + outprob[j];
            }
            x = LZERO;
            for (i = 2; i < curAc->Nq; ++i) {
                a = curAc->transp[i * Nq1 + curAc->Nq];
                y = aq[i];
                if (a > LSMALL && y > LSMALL) {
                    x = LAddDev(x, y + a);
                    /*x = log(x + y + a);*/
                }
            }
	    aq[curAc->Nq] = x;
            /* work out the exit problem for checking purpose */
        }
    }
}


/* --------------------------- Trace Flags ------------------------ */

/*  */
void SyncDev2Host(void *devPtr, void *hostPtr, size_t size) {
    hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);
}

/*  */
void SyncHost2Dev(void *hostPtr, void *devPtr, size_t size) {
    hipMemcpy(devPtr, hostPtr, size, hipMemcpyHostToDevice);	
}

/*  */
void DevDispose(void *devPtr, size_t size) {
    hipFree(devPtr);
    GPUMemUsed -= size;
}

/*  */
void DevNew(void **devAddr, size_t size) {
    hipMalloc(devAddr, size);
    GPUMemUsed += size;
}

/*  */
void ShowGPUMemUsage(void) {
    printf("%dMB Memory Used on GPU %d\n", GPUMemUsed / 1048576, GPUDevId);
}

/*  */
void SetNSegmentCUDA(NFloat val, NFloat *segPtr, int segLen) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "SetNSegmentCUDA: Block number exceeds the maximum");
    HKern_SetNSegment<<<nBlocks, THREADPERBLOCK>>>(val, segPtr, segLen);
}

/*  */
void ClearNSegmentCUDA(NFloat *segPtr, int segLen) {
    int nBlocks;
    hipError_t status;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ClearNSegmentCUDA: Block number exceeds the maximum");
    /*HKern_SetNSegment<<<nBlocks, THREADPERBLOCK>>>(0, segPtr, segLen);*/
    status = hipMemset(segPtr, 0, segLen * sizeof(NFloat));
    if (status != hipSuccess) {
        HError(9999, "ClearNSegmentCUDA: hipMemset funtion failed");
    }
    /*hipDeviceSynchronize();*/
}


/*  */
void CopyNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat *dstPtr) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDcopy(handle, segLen, srcPtr, 1, dstPtr, 1);
#else
    status = hipblasScopy(handle, segLen, srcPtr, 1, dstPtr, 1);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(9999, "CopyNSegmentCUDA: CUBLAS library copy function failed");
    }
}

/*  */
void AddNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat *dstPtr) {
    hipblasStatus_t status;
    const NFloat alpha = 1.0;

#ifdef DOUBLEANN
    status = hipblasDaxpy(handle, segLen, &alpha, srcPtr, 1, dstPtr, 1);
#else
    status = hipblasSaxpy(handle, segLen, &alpha, srcPtr, 1, dstPtr, 1);
#endif

    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(9999, "AddNSegmentCUDA: CUBLAS library copy function failed");
    }
}

/* cz277 - l2 fix */
void AddScaledNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat scale, NFloat *dstPtr) {
    hipblasStatus_t status;
    const NFloat alpha = scale;

#ifdef DOUBLEANN
    status = hipblasDaxpy(handle, segLen, &alpha, srcPtr, 1, dstPtr, 1);
#else
    status = hipblasSaxpy(handle, segLen, &alpha, srcPtr, 1, dstPtr, 1);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(9999, "AddScaledNSegmentCUDA: CUBLAS library copy function failed");
    }
    
}

/*  */
void ScaleNSegmentCUDA(int segLen, NFloat scale, NFloat *valPtr) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDscal(handle, segLen, &scale, valPtr, 1);
#else
    status = hipblasSscal(handle, segLen, &scale, valPtr, 1);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(9999, "ScaleNSegmentCUDA: CUBLAS library copy function failed");
    }
}

/*  */
void ScaledSelfAddNSegmentCUDA(NFloat *rhPtr, int segLen, NFloat scale, NFloat *lhPtr) {
    int nBlocks;
    
    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ScaledSelfAddNSegmentCUDA: Block number exceeds the maximum");
    HKern_ScaledSelfAddNSegment<<<nBlocks, THREADPERBLOCK>>>(rhPtr, segLen, scale, lhPtr);
}

/*  */
void DupNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat *dstPtr, int times) {
    int nBlocks;

    nBlocks = CEIL(segLen * times, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "DupNSegmentCUDA: Block number exceeds the maximum");
    HKern_DupNSegment<<<nBlocks, THREADPERBLOCK>>>(srcPtr, segLen, dstPtr, times);
}

/*  */
void SubNSegmentCUDA(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int nBlocks;
  
    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "SubNSegmentCUDA: Block number exceeds the maximum");
    HKern_SubNSegment<<<nBlocks, THREADPERBLOCK>>>(lhPtr, rhPtr, segLen, resPtr);
}

/*  */
void MulNSegmentCUDA(NFloat *lhPtr, NFloat *rhPtr, int segLen, NFloat *resPtr) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "MulNSegmentCUDA: Block number exceeds the maximum");
    HKern_MulNSegment<<<nBlocks, THREADPERBLOCK>>>(lhPtr, rhPtr, segLen, resPtr);
}

/* cz277 - pact */
void ApplyAffineActCUDA(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);    
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyAffineActCUDA: Block number exceeds the maximum");
    HKern_ApplyAffineAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, scalePtr, shiftPtr, dstPtr);
}

/* cz277 - pact */
void ApplyDAffineActCUDA(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDAffineActCUDA: Block number exceeds the maximum");
    HKern_ApplyDAffineAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, scalePtr, shiftPtr, dstPtr);
}


/* cz277 - pact */
void ApplyTrAffineActCUDA(NFloat *errPtr, NFloat *actPtr, int row, int col, NFloat *scalePtr, NFloat *shiftPtr, Boolean accFlag, NFloat *dScalePtr, NFloat *dShiftPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = 2 * sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyTrStdDevAffineActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrAffineAct<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, actPtr, row, col, scalePtr, shiftPtr, accFlag, dScalePtr, dShiftPtr);
}


/* cz277 - laf */
void AccMeanNSegmentCUDA(NFloat *valPtr, int row, int col, NFloat tSamp, NFloat *meanPtr) {
    int nBlocks, sBytes;
    
    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "AccMeanNSegmentCUDA: Block number exceeds the maximum");
    HKern_AccMeanNSegment<<<nBlocks, THREADPERBLOCK, sBytes>>>(valPtr, row, col, tSamp, meanPtr);
}

/* cz277 - laf */
void AccVarianceNSegmentCUDA(NFloat *valPtr, int row, int col, NFloat tSamp, NFloat *meanPtr, NFloat *varPtr) {
    int nBlocks, sBytes;
    
    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "AccVarianceNSegmentCUDA: Block number exceeds the maximum");
    HKern_AccVarianceNSegment<<<nBlocks, THREADPERBLOCK, sBytes>>>(valPtr, row, col, tSamp, meanPtr, varPtr);
}

/* cz277 - pact */
void ApplyParmReLUActCUDA(NFloat *srcPtr, int row, int col, NFloat *posPtr, NFloat *negPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyParmReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyParmReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, posPtr, negPtr, dstPtr);
}

/* cz277 - pact */
void ApplyDParmReLUActCUDA(NFloat *inpPtr, int row, int col, NFloat *posPtr, NFloat *negPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDParmReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyDParmReLUAct<<<nBlocks, THREADPERBLOCK>>>(inpPtr, row, col, posPtr, negPtr, dstPtr);
}

/* cz277 - pact */
void ApplyTrParmReLUActCUDA(NFloat *errPtr, NFloat *inpPtr, int row, int col, Boolean accFlag, NFloat *dPosPtr, NFloat *dNegPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = 2 * sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyTrParmReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrParmReLUAct<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, inpPtr, row, col, accFlag, dPosPtr, dNegPtr);
}

/* cz277 - pact */
void ApplyPReLUActCUDA(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyPReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyPReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, scalePtr, dstPtr);
}

/* cz277 - pact */
void ApplyDPReLUActCUDA(NFloat *srcPtr, int row, int col, NFloat *scalePtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDPReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyDPReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, scalePtr, dstPtr);
}

/* cz277 - pact */
void ApplyTrPReLUActCUDA(NFloat *errPtr, NFloat *srcPtr, int row, int col, NFloat *scalePtr, Boolean accFlag, NFloat *dScalePtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyTrPReLUActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrPReLUAct<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, srcPtr, row, col, scalePtr, accFlag, dScalePtr);
}

/*  */
void ApplyReLUActCUDA(NFloat *srcPtr, int len, NFloat scale, NFloat *dstPtr) {
    int nBlocks;
    
    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyReLActCUDA: Block number exceeds the maximum");
    HKern_ApplyReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, scale, dstPtr);
}

/*  */
void ApplyDReLUActCUDA(NFloat *srcPtr, int len, NFloat scale, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDReLActCUDA: Block number exceeds the maximum");
    HKern_ApplyDReLUAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, scale, dstPtr);
}

/*  */
void ApplyDLinearActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDLinearActCUDA: Block number exceeds the maximum");
    HKern_ApplyDLinearAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

void ApplyLHUCSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *rolePtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyLHUCSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyLHUCSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, rolePtr, dstPtr);
}

void ApplyDLHUCSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *rolePtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDLHUCSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyDLHUCSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, rolePtr, dstPtr);
}

void ApplyTrLHUCSigmoidActCUDA(NFloat *errPtr, NFloat *actPtr, int row, int col, NFloat *rolePtr, Boolean accFlag, NFloat *dRolePtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyTrLHUCSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrLHUCSigmoidActCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, actPtr, row, col, rolePtr, accFlag, dRolePtr); 
}

void ApplyPSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyPSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyPSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, etaPtr, dstPtr);
}

void ApplyDPSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDPSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyDPSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, etaPtr, dstPtr);
}

void ApplyTrPSigmoidActCUDA(NFloat *errPtr, NFloat *srcPtr, NFloat *etaPtr, int row, int col, Boolean accFlag, NFloat *dEtaPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyTrPSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrPSigmoidActCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, srcPtr, etaPtr, row, col, accFlag, dEtaPtr);
}


void ApplyParmSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyParmSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyParmSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, etaPtr, gammaPtr, thetaPtr, dstPtr);
}

void ApplyDParmSigmoidActCUDA(NFloat *srcPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row * col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDParmSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyDParmSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, etaPtr, gammaPtr, thetaPtr, dstPtr);
}

void ApplyTrParmSigmoidActCUDA(NFloat *errPtr, NFloat *inpPtr, int row, int col, NFloat *etaPtr, NFloat *gammaPtr, NFloat *thetaPtr, Boolean accFlag, NFloat *dEtaPtr, NFloat *dGammaPtr, NFloat *dThetaPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = 3 * sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyTrParmSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyTrParmSigmoidActCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(errPtr, inpPtr, row, col, etaPtr, gammaPtr, thetaPtr, accFlag, dEtaPtr, dGammaPtr, dThetaPtr);
}


/*  */
void ApplySigmoidActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplySigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplySigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplyDSigmoidActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDSigmoidActCUDA: Block number exceeds the maximum");
    HKern_ApplyDSigmoidAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplyTanHActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyTanHActCUDA: Block number exceeds the maximum");
    HKern_ApplyTanHAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplyDTanHActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyDTanHActCUDA: Block number exceeds the maximum");
    HKern_ApplyDTanHAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}


/*  */
void ApplyRedSoftmaxActCUDA(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int nBlocks, sBytes;

    nBlocks = row;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyRedSoftmaxActCUDA: Block number exceeds the maximum");
    HKern_ApplyRedSoftmaxAct<<<nBlocks, THREADPERBLOCK, sBytes>>>(srcPtr, row, col, dstPtr);
}

/*  */
void ApplySoftmaxActCUDA(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(row, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplySoftmaxActCUDA: Block number exceeds the maximum");
    HKern_ApplySoftmaxAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, dstPtr);
}

/*  */
void ApplySoftReLActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;
 
    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplySoftReLActCUDA: Block number exceeds the maximum");
    HKern_ApplySoftReLAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplyDSoftReLActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplySoftReLActCUDA: Block number exceeds the maximum");
    HKern_ApplyDSoftReLAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);
}

/*  */
void ApplySoftSignActCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplySoftSignActCUDA: Block number exceeds the maximum");
    HKern_ApplySoftSignAct<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);    
}

/*  */
void ApplyLogTransCUDA(NFloat *srcPtr, int len, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ApplyLogTransCUDA: Block number exceeds the maximum");
    HKern_ApplyLogTrans<<<nBlocks, THREADPERBLOCK>>>(srcPtr, len, dstPtr);    
}

/*  */
void RedSumNMatrixByColCUDA(NFloat *srcPtr, int row, int col, Boolean accFlag, NFloat *dstPtr) {
    int nBlocks, sBytes;

    nBlocks = col;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "RedSumNMatrixByColCUDA: Block number exceeds the maximum");
    HKern_RedSumNMatrixByColCUDA<<<nBlocks, THREADPERBLOCK, sBytes>>>(srcPtr, row, col, accFlag, dstPtr);
}

/*  */
void SumNMatrixByColCUDA(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(col, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "SumNMatrixByColCUDA: Block number exceeds the maximum");
    HKern_SumNMatrixByCol<<<nBlocks, THREADPERBLOCK>>>(srcPtr, row, col, dstPtr);
}

/*  */
void SquaredNSegmentCUDA(NFloat *srcPtr, int segLen, NFloat *dstPtr) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "SquaredNSegmentCUDA: Block number exceeds the maximum");
    HKern_SquaredNSegment<<<nBlocks, THREADPERBLOCK>>>(srcPtr, segLen, dstPtr);
}

/*  */
void CompAdaGradNSegmentCUDA(NFloat eta, int K, int segLen, NFloat *ssgSeg, NFloat *nlrSeg) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "CompAdaGradNSegmentCUDA: Block number exceeds the maximum");
    HKern_CompAdaGradNSegment<<<nBlocks, THREADPERBLOCK>>>(eta, K, segLen, ssgSeg, nlrSeg);
}

/*  */
void HNBlasNNgemmCUDA(int m, int n, int k, NFloat alpha, NFloat *A, NFloat *B, NFloat beta, NFloat *C) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, m, B, k, &beta, C, m);
#else
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, A, m, B, k, &beta, C, m);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(9999, "HNBlasNNgemmCUDA: CUBLAS library gemm function failed");
    }
}

/*  */
void HNBlasNTgemmCUDA(int m, int n, int k, NFloat alpha, NFloat *A, NFloat *B, NFloat beta, NFloat *C) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, m, B, n, &beta, C, m);
#else
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, m, B, n, &beta, C, m);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(9999, "HNBlasNTgemmCUDA: CUBLAS library gemm function failed");
    }
}

/*  */
void HNBlasTNgemmCUDA(int m, int n, int k, NFloat alpha, NFloat *A, NFloat *B, NFloat beta, NFloat *C) {
    hipblasStatus_t status;

#ifdef DOUBLEANN
    status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, C, m);
#else
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, k, B, k, &beta, C, m);
#endif
    if (status != HIPBLAS_STATUS_SUCCESS) {
        HError(9999, "HNBlasTNgemmCUDA: CUBLAS library gemm function failed");
    }
}

/*  */
void CalXENTCriterionCUDA(NFloat *refPtr, NFloat *hypPtr, int segLen, NFloat *crtPtr) {
    HKern_CalXENTCriterionCUDA<<<1, THREADPERBLOCK>>>(refPtr, hypPtr, segLen, crtPtr);
}

/*  */
void CalMMSECriterionCUDA(NFloat *refPtr, NFloat *hypPtr, int segLen, NFloat *crtPtr) {
    HKern_CalMMSECriterionCUDA<<<1, THREADPERBLOCK>>>(refPtr, hypPtr, segLen, crtPtr);
}

/*  */
void AddNSegmentTargetPenCUDA(NFloat *srcSeg, NFloat *penSeg, int row, int col, NFloat *dstSeg) {
    int nBlocks, size;

    size = row * col;
    nBlocks = CEIL(size, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "AddNVectorTargetPenCUDA: Block number exceeds the maximum");

    HKern_AddSegmentTargetPen<<<nBlocks, THREADPERBLOCK>>>(srcSeg, penSeg, row, col, dstSeg);
}

void FindMaxElementCUDA(NFloat *srcPtr, int row, int col, NFloat *dstPtr) {
    int nBlocks, sBytes;

    nBlocks = row;
    sBytes = 2 * sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "FindMaxElementCUDA: Block number exceeds the maximum");
    HKern_RedMaxElementIndex<<<nBlocks, THREADPERBLOCK, sBytes>>>(srcPtr, row, col, dstPtr);
}

/*  */
/*void SubNSegmentByConstCUDA(NFloat *srcSeg, int segLen, NFloat constVal, NFloat *dstSeg) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK); 
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "SubNSegmentByConstCUDA: Block number exceeds the maximum");

    HKern_SubNSegmentByConst<<<nBlocks, THREADPERBLOCK>>>(srcSeg, segLen, constVal, dstSeg);
}*/

/* cz277 - semi */
/*  */
void ShiftNSegmentValsCUDA(NFloat *srcSeg, int segLen, NFloat shiftVal, NFloat *dstSeg) {
    int nBlocks;

    nBlocks = CEIL(segLen, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ShiftNSegmentValsCUDA: Block number exceeds the maximum");

    HKern_ShiftNSegmentVals<<<nBlocks, THREADPERBLOCK>>>(srcSeg, segLen, shiftVal, dstSeg);
}

/* cz277 - 1007 */
void CopyPartialNSegmentCUDA(int minRow, int minCol, NFloat *srcPtr, int srcCol, NFloat *dstPtr, int dstCol) {
    int len, nBlocks;

    len = minRow * minCol;
    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "CopyPartialNSegmentCUDA: Block number exceeds the maximum");
    HKern_CopyPartialNSegment<<<nBlocks, THREADPERBLOCK>>>(minRow, minCol, srcPtr, srcCol, dstPtr, dstCol);
}

/* --------------------------- HFBLat funcs ------------------------ */

/* cz277 - cuda fblat */
void SetModelBetaPlusCUDA(int T, NMatrix *llhMat, int *qLo, int *qHi, int Q, float probScale, AcousticDev *acList) {
    int nBlocks;

    /* t in [1 ... T]; q in [1 ... Q] */
    nBlocks = CEIL(T * Q, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "SetModelBetaPlusCUDA: Block number exceeds the maximum");
    /* setotprob */
    HKern_Setotprob4q<<<nBlocks, THREADPERBLOCK>>>(T, llhMat->devElems, llhMat->colNum, qLo, qHi, Q, probScale, acList);
    /* set model beta plus */
    nBlocks = CEIL(Q, THREADPERBLOCK);
    HKern_SetModelPlus<<<nBlocks, THREADPERBLOCK>>>(Q, acList);

} 


/* cz277 - cuda fblat */
void ZeroAlphasCUDA(int T, int Q, AcousticDev *acList) {
    int nBlocks;

    nBlocks = CEIL(T * Q, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "ZeroAlphasCUDA: Block number exceeds the maximum");
    HKern_ZeroAlphas<<<nBlocks, THREADPERBLOCK>>>(T, Q, acList);
}


/* cz277 - cuda fblat */
void StepAlphaCUDA(int Q, AcousticDev *acList) {
    int nBlocks;

    nBlocks = CEIL(Q, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "StepAlphaCUDA: Block number exceeds the maximum");
    HKern_StepAlpha<<<nBlocks, THREADPERBLOCK>>>(Q, acList);
}

/* cz277 - gradlim */
void ClipNSegmentValsCUDA(NFloat* srcSeg, int len, NFloat upperLim, NFloat lowerLim, NFloat *dstSeg) {
    int nBlocks;

    nBlocks = CEIL(len, THREADPERBLOCK);
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "LimitNSegmentValsCUDA: Block number exceeds the maximum");
    HKern_ClipNSegmentVals<<<nBlocks, THREADPERBLOCK>>>(srcSeg, len, upperLim, lowerLim, dstSeg);
}

/* cz277 - max norm */
void CalExtNMatrixL2NormCUDA(NFloat *matPtr, NFloat *vecPtr, int row, int col, NFloat *alphas) {
    int nBlocks, sBytes;
  
    nBlocks = row;
    sBytes = sizeof(NFloat) * THREADPERBLOCK;
    if (nBlocks > MAXBLOCKNUM)
        HError(9999, "CalExtNMatrixL2NormCUDA: Block number exceeds the maximum");
    HKern_RedCalExtNMatrixL2Norm<<<nBlocks, THREADPERBLOCK, sBytes>>>(matPtr, vecPtr, row, col, alphas);
    nBlocks = 1;
    HKern_RedMaxElementValue<<<nBlocks, THREADPERBLOCK, sBytes>>>(alphas, 1, row, alphas);
}


/* --------------------------- Trace Flags ------------------------ */


#ifdef __cplusplus
}
#endif


